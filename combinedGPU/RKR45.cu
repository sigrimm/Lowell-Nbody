
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <string.h>

#define dayUnit 0.01720209895
//#define dayUnit 0.01720209894846
//#define dayUnit 1.0

//#define def_c 10065.3201686
#define def_c 10065.320121
#define def_AU 149597870700.0           //AU in m

//constant memory
__constant__ double  a_c[6 * 6];
__constant__ double  b_c[6];
__constant__ double  c_c[6];


// --------------------------------
//barycentric coordinates
__device__ __host__ void acc(double *m, double *x, double *y, double *z, double &ax, double &ay, double &az, int i, int j){

	double rx = x[j] - x[i];
	double ry = y[j] - y[i];
	double rz = z[j] - z[i];
	double rsq = rx * rx + ry * ry + rz * rz;
	double r = sqrt(rsq);

	double s = m[j] / (r * rsq);
	
	ax += s * rx;
	ay += s * ry;
	az += s * rz;
}
// ---------------------------------

// --------------------------------
//heliocentric coordinates
//sun part
__host__ void accS(double *m, double *x, double *y, double *z, double &ax, double &ay, double &az, int i){

	if(i > 0){
		double rx = -x[i];
		double ry = -y[i];
		double rz = -z[i];
		double rsq = rx * rx + ry * ry + rz * rz;
		double r = sqrt(rsq);

		double s = (m[0] + m[i]) / (r * rsq);

		ax += s * rx;
		ay += s * ry;
		az += s * rz;
	}
}
__device__ void accS_device(double mu, double xi, double yi, double zi, double &ax, double &ay, double &az){

	double rx = -xi;
	double ry = -yi;
	double rz = -zi;
	double rsq = rx * rx + ry * ry + rz * rz;
	double r = sqrt(rsq);

	double s = mu / (r * rsq);

	ax += s * rx;
	ay += s * ry;
	az += s * rz;
}
//planet part
__host__ void accP(double *m, double *x, double *y, double *z, double &ax, double &ay, double &az, int i, int j){

	if(i != j){
		double rx = x[j] - x[i];
		double ry = y[j] - y[i];
		double rz = z[j] - z[i];
		double rsq = rx * rx + ry * ry + rz * rz;
		double r = sqrt(rsq);

		double s = m[j] / (r * rsq);

		ax += s * rx;
		ay += s * ry;
		az += s * rz;
	}
}
__device__ void accP_device(double mj, double xj, double yj, double zj, double xi, double yi, double zi, double &ax, double &ay, double &az){

	double rx = xj - xi;
	double ry = yj - yi;
	double rz = zj - zi;
	double rsq = rx * rx + ry * ry + rz * rz;
	double r = sqrt(rsq);

	double s = mj / (r * rsq);

	ax += s * rx;
	ay += s * ry;
	az += s * rz;
}
//planet part 2
__host__ void accP2(double *m, double *x, double *y, double *z, double &ax, double &ay, double &az, int i, int j){

	if(i != j){
		double rx = -x[j];
		double ry = -y[j];
		double rz = -z[j];
		double rsq = rx * rx + ry * ry + rz * rz;
		double r = sqrt(rsq);

		double s = m[j] / (r * rsq);
	
		ax += s * rx;
		ay += s * ry;
		az += s * rz;
	}
}
__device__ void accP2_device(double mj, double xj, double yj, double zj, double &ax, double &ay, double &az){

	double rx = -xj;
	double ry = -yj;
	double rz = -zj;
	double rsq = rx * rx + ry * ry + rz * rz;
	double r = sqrt(rsq);

	double s = mj / (r * rsq);
	ax += s * rx;
	ay += s * ry;
	az += s * rz;
}
// --------------------------------------


//Sitarski 1982, Isotropic equation 5, heliocentric
//modified k2 to dayUnit
//should be equivalent to the Quinn et all function, assuming m[0] = 1.0
//heliocentric
__device__ __host__ void acchGR2(double xi, double yi, double zi, double vxi, double vyi, double vzi, double &ax, double &ay, double &az){
	
	double c2 = def_c * def_c;

	double rsq = xi * xi + yi * yi + zi * zi;
	double r = sqrt(rsq);
	double vsq = vxi * vxi + vyi * vyi + vzi * vzi;

	double rv = xi * vxi + yi * vyi + zi * vzi;

	double f1 = 1.0 / (r * rsq * c2);
	double t1 = 4.0 / r;
	double t2 = -vsq;
	double t3 = 4.0 * rv;
//printf("a %d %.20g %.20g %.20g\n", i, ax, ay, az);

//printf("A %d %.20g %.20g %.20g %.20g %.20g %.20g\n", i, xi, yi, zi, vxi, vyi, vzi);
//printf("B %d %.20g %.20g %.20g %.20g\n", i, f1, t1, t2, t3);
//printf("C %d %.20g %.20g %.20g %.20g\n", i, t1 + t2, (t1 + t2) * xi, ((t1 + t2) * xi + t3 * vxi), f1 * ((t1 + t2) * xi + t3 * vxi));

	ax += f1 * ((t1 + t2) * xi + t3 * vxi);
	ay += f1 * ((t1 + t2) * yi + t3 * vyi);
	az += f1 * ((t1 + t2) * zi + t3 * vzi);
//printf("D %d %.20g %.20g %.20g\n", i, ax, ay, az);

}

//Neville-Aitken interpolation
__host__ void interpolate(int Ninterpolate, double *xp, double *yp, double *zp, double *timep, double time, double *xt, double *yt, double *zt, int p){


	double Px[Ninterpolate][Ninterpolate];
	double Py[Ninterpolate][Ninterpolate];
	double Pz[Ninterpolate][Ninterpolate];
	double tn[Ninterpolate];

	for(int i = 0; i < Ninterpolate; ++i){
		Px[0][i] = xp[p * Ninterpolate + i];
		Py[0][i] = yp[p * Ninterpolate + i];
		Pz[0][i] = zp[p * Ninterpolate + i];
		tn[i] = timep[p * Ninterpolate + i];

//printf("interpolate %d %d %.20g %.20g %.20g\n", p, i, time, tn[i], P[0][i]);
	}

	for(int j = 1; j < Ninterpolate; ++j){
//printf("****\n");
		for(int i = 0; i < Ninterpolate - j; ++i){
			Px[j][i] = ((time - tn[i+j]) * Px[j-1][i] + (tn[i] - time) * Px[j-1][i+1]) / (tn[i] - tn[i+j]);
			Py[j][i] = ((time - tn[i+j]) * Py[j-1][i] + (tn[i] - time) * Py[j-1][i+1]) / (tn[i] - tn[i+j]);
			Pz[j][i] = ((time - tn[i+j]) * Pz[j-1][i] + (tn[i] - time) * Pz[j-1][i+1]) / (tn[i] - tn[i+j]);
//printf("%d %d %g %g %g %g %.20g\n", i, i+j, tn[i], tn[i+j], P[j-1][i], P[j-1][i+1], P[j][i]);

		}
	}
	xt[p] = Px[Ninterpolate-1][0];
	yt[p] = Py[Ninterpolate-1][0];
	zt[p] = Pz[Ninterpolate-1][0];
//printf("interpolate %.20g %d %.20g %.20g %.20g\n", time, p, xt[p], yt[p], zt[p]);

}
__host__ void interpolate2(int Ninterpolate, double *xp, double *yp, double *zp, double *timep, double time, double *xt, double *yt, double *zt, int p){


	//p is the particle index

	double Cx[Ninterpolate];
	double Cy[Ninterpolate];
	double Cz[Ninterpolate];

	double Dx[Ninterpolate];
	double Dy[Ninterpolate];
	double Dz[Ninterpolate];

	double tn[Ninterpolate];

	for(int i = 0; i < Ninterpolate; ++i){
		Cx[i] = xp[p * Ninterpolate + i];
		Cy[i] = yp[p * Ninterpolate + i];
		Cz[i] = zp[p * Ninterpolate + i];

		Dx[i] = Cx[i];		
		Dy[i] = Cy[i];		
		Dz[i] = Cz[i];		

		tn[i] = timep[p * Ninterpolate + i];

//printf("interpolate %d %d %.20g %.20g %.20g\n", p, i, time, tn[i], Cx[i]);
	}

	//initialize with closest solution
	//Assume that the closest solution is in the middle

	int ii = Ninterpolate / 2 - 1;
	xt[p] = Cx[ii];
	yt[p] = Cy[ii];
	zt[p] = Cz[ii];

	--ii;

	for(int j = 1; j < Ninterpolate; ++j){
//printf("**** %d %d %g\n", j, ii, xt[p]);
		for(int i = 0; i < Ninterpolate - j; ++i){

			double dtn0 = tn[i] - time;
			double dtn1 = tn[i + j] - time;
			double dtn = tn[i] - tn[i + j];

			double dPx = (Cx[i + 1] - Dx[i]) / dtn;
			double dPy = (Cy[i + 1] - Dy[i]) / dtn;
			double dPz = (Cz[i + 1] - Dz[i]) / dtn;

			Dx[i] = dtn1 * dPx;
			Dy[i] = dtn1 * dPy;
			Dz[i] = dtn1 * dPz;

			Cx[i] = dtn0 * dPx;
			Cy[i] = dtn0 * dPy;
			Cz[i] = dtn0 * dPz;

	
//printf("%d %d %g %g %g %g %.20g\n", i, i+j, tn[i], tn[i+j], P[j-1][i], P[j-1][i+1], P[j][i]);

		}

		if(2 * ii < Ninterpolate - j){
			xt[p] += Cx[ii + 1];
			yt[p] += Cy[ii + 1];
			zt[p] += Cz[ii + 1];
		}
		else{
			xt[p] += Dx[ii];
			yt[p] += Dy[ii];
			zt[p] += Dz[ii];
			--ii;
		}

	}
//printf("interpolate %.20g %d %.20g %.20g %.20g\n", time, p, xt[p], yt[p], zt[p]);

}


template <int Ninterpolate>
__global__ void interpolate_kernel(int Nperturbers, double *xp, double *yp, double *zp, double *timep, double time, double *xt, double *yt, double *zt){

	int pid = blockIdx.x;	//perturber index, Nperturbers
	int idx = threadIdx.x;

	if(pid < Nperturbers){

		__shared__ double Px_s[Ninterpolate][Ninterpolate];
		__shared__ double Py_s[Ninterpolate][Ninterpolate];
		__shared__ double Pz_s[Ninterpolate][Ninterpolate];
		__shared__ double tn_s[Ninterpolate];

		if(idx < Ninterpolate){
			Px_s[0][idx] = xp[pid * Ninterpolate + idx];
			Py_s[0][idx] = yp[pid * Ninterpolate + idx];
			Pz_s[0][idx] = zp[pid * Ninterpolate + idx];
			tn_s[idx] = timep[pid * Ninterpolate + idx];

//printf("interpolate %d %d %.20g %.20g %.20g\n", p, i, time, tn_s[i], Px_s[0][i]);
		}
		__syncthreads();

		for(int j = 1; j < Ninterpolate; ++j){
//printf("****\n");
			if(idx < Ninterpolate - j){
				Px_s[j][idx] = ((time - tn_s[idx + j]) * Px_s[j-1][idx] + (tn_s[idx] - time) * Px_s[j-1][idx + 1]) / (tn_s[idx] - tn_s[idx + j]);
				Py_s[j][idx] = ((time - tn_s[idx + j]) * Py_s[j-1][idx] + (tn_s[idx] - time) * Py_s[j-1][idx + 1]) / (tn_s[idx] - tn_s[idx + j]);
				Pz_s[j][idx] = ((time - tn_s[idx + j]) * Pz_s[j-1][idx] + (tn_s[idx] - time) * Pz_s[j-1][idx + 1]) / (tn_s[idx] - tn_s[idx + j]);
//printf("%d %d %g %g %g %g %.20g\n", idx, idx+j, tn_s[idx], tn_s[idx + j], Px_s[j-1][idx], Px_s[j-1][idx + 1], Px_s[j][idx]);
			}
			__syncthreads();
		}

		if(idx == 0){
			xt[pid] = Px_s[Ninterpolate-1][0];
			yt[pid] = Py_s[Ninterpolate-1][0];
			zt[pid] = Pz_s[Ninterpolate-1][0];
		}
//printf("interpolate %.20g %d %.20g %.20g %.20g\n", time, pid, xt[pid], yt[pid], zt[pid]);

	}
}


//A1, A2 and A3 terms for asteroids on heliocentric coordinates
void NonGrav(double *x, double *y, double *z, double *vx, double *vy, double *vz, double &ax, double &ay, double &az, double *A1, double *A2, double *A3, double *ALN, double *NK, double *NM, double *Nn, double *R0, int i){

	double rsq = x[i] * x[i] + y[i] * y[i] + z[i] * z[i];
	double r = sqrt(rsq);

	//angular momenrum h = r x v
	double hx = y[i] * vz[i] - z[i] * vy[i];
	double hy =-x[i] * vz[i] + z[i] * vx[i];
	double hz = x[i] * vy[i] - y[i] * vx[i];

	double hsq = hx * hx + hy * hy + hz * hz;
	double h = sqrt(hsq);

	//Transverse velocity t = h x r
	double tx = hy * z[i] - hz * y[i];
	double ty =-hx * z[i] + hz * x[i];
	double tz = hx * y[i] - hy * x[i];

	double tsq = tx * tx + ty * ty + tz * tz;
	double t = sqrt(tsq);

	double gr = 1.0 / rsq;	//only valid for asteroids, not for comets 
/*
	double rr = r / R0[i];
	double g1 = pow(rr, -NM[i]);
	double g2 = pow(rr, Nn[i]);
	double g3 = pow(1.0 + g2, -NK[i]);
	double gr = ALN[i] * g1 * g3;

printf("gr %.20g %.20g\n", gr1, gr);
*/

	double f1 = A1[i] * gr / r;
	double f2 = A2[i] * gr / t;
	double f3 = A3[i] * gr / h;
	
	
	ax += f1 * x[i] + f2 * tx + f3 * hx;
	ay += f1 * y[i] + f2 * ty + f3 * hy;
	az += f1 * z[i] + f2 * tz + f3 * hz;
//printf("NonGrav %d %.20g %.20g %.20g\n", i, (f1 * x[i] + f2 * tx + f3 * hx) * dayUnit * dayUnit, (f1 * y[i] + f2 * ty + f3 * hy) * dayUnit * dayUnit, (f1 * z[i] + f2 * tz + f3 * hz) * dayUnit * dayUnit);

}

//J2 perturbation from Earth
void J2(double *m, double *x, double *y, double *z, double &ax, double &ay, double &az, int i){

	double J2E = 0.00108262545; // J2 Earth from DE 430
	double RE = 6378136.3; // Earth radius in m from DE 430

	//double J2E =  1.08263e-3; //  1.08262668e-3;
	//double RE = 6371.009; // Earth radius in km
	//double muE = 398600.44 // in km^3 /s^2	G * mEarth

	RE /= def_AU;	//Earth radius in AU

	int iE = 3; 	//index of Earth

	double muE = m[iE];

	double xE = x[i] - x[iE];
	double yE = y[i] - y[iE];
	double zE = z[i] - z[iE];

	double rsq = xE * xE + yE * yE + zE * zE;
	double r = sqrt(rsq);
	double r5 = rsq * rsq * r;

	double t1 = 3.0 * J2E * muE * RE * RE / (2.0 * r5);
	double t2 = 5.0 * zE * zE / rsq;

//printf("rm %.20g %.20g %.20g\n", RE, muE, t1);

	double tx = t1 * (t2 - 1.0) * xE;
	double ty = t1 * (t2 - 1.0) * yE;
	double tz = t1 * (t2 - 3.0) * zE;
	
	ax += tx;
	ay += ty;
	az += tz;
 
//printf("J2 %d %.20g %.20g %.20g | %.20g %.20g %.20g\n", i, tx * dayUnit * dayUnit, ty * dayUnit * dayUnit, tz * dayUnit * dayUnit, xE, yE, zE); 


}

__host__ void stageStep(double *m, double *xt, double *yt, double *zt, double *vxt, double *vyt, double *vzt, double *kx, double *ky, double *kz, double *kvx, double *kvy, double *kvz, int S, int i, int Nperturbers, int N, int useHelio, int GR){

	kx[i + S * N] = vxt[i];
	ky[i + S * N] = vyt[i];
	kz[i + S * N] = vzt[i];

	double ax = 0.0;
	double ay = 0.0;
	double az = 0.0;

	if(useHelio == 0){
		for(int j = Nperturbers-1; j >= 0; --j){
			accP(m, xt, yt, zt, ax, ay, az, i, j);
		}
	}
	else{
		for(int j = Nperturbers-1; j >= 1; --j){
			accP(m, xt, yt, zt, ax, ay, az, i, j);
//if(i == 27) printf("Nij %d %d %.20g %.20g %.20g\n", i, j, ax * dayUnit * dayUnit, ay * dayUnit * dayUnit, az * dayUnit * dayUnit);
		}
		accS(m, xt, yt, zt, ax, ay, az, i);
//if(i == 27) printf("Nij %d %d %.20g %.20g %.20g\n", i, 0, ax * dayUnit * dayUnit, ay * dayUnit * dayUnit, az * dayUnit * dayUnit);
//printf("N0 %d %.20g %.20g %.20g\n", i, ax * dayUnit * dayUnit, ay * dayUnit * dayUnit, az * dayUnit * dayUnit);
		for(int j = Nperturbers-1; j >= 1; --j){
			accP2(m, xt, yt, zt, ax, ay, az, i, j);
		}
//if(i == 27) printf("Np %d %.20g %.20g %.20g %d\n", i, ax * dayUnit * dayUnit, ay * dayUnit * dayUnit, az * dayUnit * dayUnit, S);
	}


	if(GR == 2){
		acchGR2(xt[i], yt[i], zt[i], vxt[i], vyt[i], vzt[i], ax, ay, az);
	}

	//NonGrav(xt, yt, zt, vxt, vyt, vzt, ax, ay, az, A1, A2, A3, ALN, NK, NM, Nn, R0, i);
	//J2(m, xt, yt, zt, ax, ay, az, i);

	kvx[i + S * N] = ax;
	kvy[i + S * N] = ay;
	kvz[i + S * N] = az;
}

template < int NN >
__global__ void stageStep_kernel(double *m_d, double *x_d, double *y_d, double *z_d, double *vx_d, double *vy_d, double *vz_d, double *xt_d, double *yt_d, double *zt_d, double *vxt_d, double *vyt_d, double *vzt_d, double *kx_d, double *ky_d, double *kz_d, double *kvx_d, double *kvy_d, double *kvz_d, double dt, int S, int Nperturbers, int N, int useHelio, int GR){

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	//shared memory contains only the perturbers
	//the particle idx is stored in registers
	__shared__ double x_s[NN];
	__shared__ double y_s[NN];
	__shared__ double z_s[NN];
	__shared__ double m_s[NN];


	if(threadIdx.x < Nperturbers){
		if(S == 0){
			x_s[threadIdx.x] = x_d[idx];
			y_s[threadIdx.x] = y_d[idx];
			z_s[threadIdx.x] = z_d[idx];
			m_s[threadIdx.x] = m_d[idx];
		}
		else{
			x_s[threadIdx.x] = xt_d[idx];
			y_s[threadIdx.x] = yt_d[idx];
			z_s[threadIdx.x] = zt_d[idx];
			m_s[threadIdx.x] = m_d[idx];
		}
	}
	__syncthreads();

	if(idx >= Nperturbers && idx < N){


// ***********************
		//update
		double xi = x_d[idx];
		double yi = y_d[idx];
		double zi = z_d[idx];
		double vxi = vx_d[idx];
		double vyi = vy_d[idx];
		double vzi = vz_d[idx];
		double mi = m_d[idx];

		for(int s = 0; s < S; ++s){
			double aa = a_c[S * 6 + s];
			xi  += dt * aa * kx_d[idx + s * N];
			yi  += dt * aa * ky_d[idx + s * N];
			zi  += dt * aa * kz_d[idx + s * N];
			vxi += dt * aa * kvx_d[idx + s * N];
			vyi += dt * aa * kvy_d[idx + s * N];
			vzi += dt * aa * kvz_d[idx + s * N];
		}
// *****************************

		kx_d[idx + S * N] = vxi;
		ky_d[idx + S * N] = vyi;
		kz_d[idx + S * N] = vzi;

		double ax = 0.0;
		double ay = 0.0;
		double az = 0.0;

		if(useHelio == 0){
			for(int j = Nperturbers-1; j >= 0; --j){
				accP_device(m_s[j], x_s[j], y_s[j], z_s[j], xi, yi, zi, ax, ay, az);
			}
		}
		else{
			for(int j = Nperturbers-1; j >= 1; --j){
				accP_device(m_s[j], x_s[j], y_s[j], z_s[j], xi, yi, zi, ax, ay, az);
//printf("Nij %d %d %.20g %.20g %.20g\n", idx, j, ax * dayUnit * dayUnit, ay * dayUnit * dayUnit, az * dayUnit * dayUnit);
			}
			accS_device(m_s[0] + mi, xi, yi, zi, ax, ay, az);

//printf("N0 %d %.20g %.20g %.20g\n", idx, ax * dayUnit * dayUnit, ay * dayUnit * dayUnit, az * dayUnit * dayUnit);
			for(int j = Nperturbers-1; j >= 1; --j){
				accP2_device(m_s[j], x_s[j], y_s[j], z_s[j], ax, ay, az);
//printf("Npi %d %d %.20g %.20g %.20g %d\n", idx, j, ax * dayUnit * dayUnit, ay * dayUnit * dayUnit, az * dayUnit * dayUnit, S);
			}
//printf("Np %d %.20g %.20g %.20g %d\n", idx, ax * dayUnit * dayUnit, ay * dayUnit * dayUnit, az * dayUnit * dayUnit, S);
		}


		if(GR == 2){
			acchGR2(xi, yi, zi, vxi, vyi, vzi, ax, ay, az);
		}

		//NonGrav(xt, yt, zt, vxt, vyt, vzt, ax, ay, az, A1, A2, A3, ALN, NK, NM, Nn, R0, idx);
		//J2(m, xt, yt, zt, ax, ay, az, idx);

		kvx_d[idx + S * N] = ax;
		kvy_d[idx + S * N] = ay;
		kvz_d[idx + S * N] = az;
	}

}

//combine stage step and interpolation kernel
template < const int NN, const int Ninterpolate >
__global__ void stageStep1_kernel(double *m_d, double *x_d, double *y_d, double *z_d, double *vx_d, double *vy_d, double *vz_d, double *xp_d, double *yp_d, double *zp_d, double *timep_d, double time, double *kx_d, double *ky_d, double *kz_d, double *kvx_d, double *kvy_d, double *kvz_d, double dt, int S, int Nperturbers, int N, int useHelio, int GR){

	int idx = threadIdx.x;
	int id = blockIdx.x * blockDim.x + idx;

	//shared memory contains only the perturbers
	//the particle idx is stored in registers
	__shared__ double x_s[NN];
	__shared__ double y_s[NN];
	__shared__ double z_s[NN];
	__shared__ double m_s[NN];

	
	//needs at least Nperturbers threads per block
	// *******************************************************
	//interpolate

	//do interpolation in every thread block
	//p is the particle index

	double Cx[Ninterpolate];
	double Cy[Ninterpolate];
	double Cz[Ninterpolate];

	double Dx[Ninterpolate];
	double Dy[Ninterpolate];
	double Dz[Ninterpolate];

	double tn[Ninterpolate];


	if(idx < Nperturbers){

		for(int i = 0; i < Ninterpolate; ++i){
			Cx[i] = xp_d[idx * Ninterpolate + i];
			Cy[i] = yp_d[idx * Ninterpolate + i];
			Cz[i] = zp_d[idx * Ninterpolate + i];

			Dx[i] = Cx[i];		
			Dy[i] = Cy[i];		
			Dz[i] = Cz[i];		

			tn[i] = timep_d[idx * Ninterpolate + i];

	//printf("interpolate %d %d %.20g %.20g %.20g\n", idx, i, time, tn[i], Cx[i]);
		}

		//initialize with closest solution
		//Assume that the closest solution is in the middle

		int ii = Ninterpolate / 2 - 1;
		x_s[idx] = Cx[ii];
		y_s[idx] = Cy[ii];
		z_s[idx] = Cz[ii];
		m_s[idx] = m_d[idx]; 

		--ii;

		for(int j = 1; j < Ninterpolate; ++j){
	//printf("**** %d %d %g\n", j, ii, x_s[idx]);
			for(int i = 0; i < Ninterpolate - j; ++i){

				double dtn0 = tn[i] - time;
				double dtn1 = tn[i + j] - time;
				double dtn = tn[i] - tn[i + j];

				double dPx = (Cx[i + 1] - Dx[i]) / dtn;
				double dPy = (Cy[i + 1] - Dy[i]) / dtn;
				double dPz = (Cz[i + 1] - Dz[i]) / dtn;

				Dx[i] = dtn1 * dPx;
				Dy[i] = dtn1 * dPy;
				Dz[i] = dtn1 * dPz;

				Cx[i] = dtn0 * dPx;
				Cy[i] = dtn0 * dPy;
				Cz[i] = dtn0 * dPz;

		
	//printf("%d %d %g %g %g %g %.20g\n", i, i+j, tn[i], tn[i+j], P[j-1][i], P[j-1][i+1], P[j][i]);

			}

			if(2 * ii < Ninterpolate - j){
				x_s[idx] += Cx[ii + 1];
				y_s[idx] += Cy[ii + 1];
				z_s[idx] += Cz[ii + 1];
			}
			else{
				x_s[idx] += Dx[ii];
				y_s[idx] += Dy[ii];
				z_s[idx] += Dz[ii];
				--ii;
			}

		}

	}

	if(id >= Nperturbers && id < N){


// ***********************
		//update
		double xi = x_d[id];
		double yi = y_d[id];
		double zi = z_d[id];
		double vxi = vx_d[id];
		double vyi = vy_d[id];
		double vzi = vz_d[id];
		double mi = m_d[id];

		for(int s = 0; s < S; ++s){
			double aa = a_c[S * 6 + s];
			xi  += dt * aa * kx_d[id + s * N];
			yi  += dt * aa * ky_d[id + s * N];
			zi  += dt * aa * kz_d[id + s * N];
			vxi += dt * aa * kvx_d[id + s * N];
			vyi += dt * aa * kvy_d[id + s * N];
			vzi += dt * aa * kvz_d[id + s * N];
		}
// *****************************

		kx_d[id + S * N] = vxi;
		ky_d[id + S * N] = vyi;
		kz_d[id + S * N] = vzi;

		double ax = 0.0;
		double ay = 0.0;
		double az = 0.0;

		if(useHelio == 0){
			for(int j = Nperturbers-1; j >= 0; --j){
				accP_device(m_s[j], x_s[j], y_s[j], z_s[j], xi, yi, zi, ax, ay, az);
			}
		}
		else{
			for(int j = Nperturbers-1; j >= 1; --j){
				accP_device(m_s[j], x_s[j], y_s[j], z_s[j], xi, yi, zi, ax, ay, az);
//printf("Nij %d %d %.20g %.20g %.20g\n", id, j, ax * dayUnit * dayUnit, ay * dayUnit * dayUnit, az * dayUnit * dayUnit);
			}
			accS_device(m_s[0] + mi, xi, yi, zi, ax, ay, az);

//printf("N0 %d %.20g %.20g %.20g\n", id, ax * dayUnit * dayUnit, ay * dayUnit * dayUnit, az * dayUnit * dayUnit);
			for(int j = Nperturbers-1; j >= 1; --j){
				accP2_device(m_s[j], x_s[j], y_s[j], z_s[j], ax, ay, az);
//printf("Npi %d %d %.20g %.20g %.20g %d\n", id, j, ax * dayUnit * dayUnit, ay * dayUnit * dayUnit, az * dayUnit * dayUnit, S);
			}
//printf("Np %d %.20g %.20g %.20g %d\n", id, ax * dayUnit * dayUnit, ay * dayUnit * dayUnit, az * dayUnit * dayUnit, S);
		}


		if(GR == 2){
			acchGR2(xi, yi, zi, vxi, vyi, vzi, ax, ay, az);
		}

		//NonGrav(xt, yt, zt, vxt, vyt, vzt, ax, ay, az, A1, A2, A3, ALN, NK, NM, Nn, R0, id);
		//J2(m, xt, yt, zt, ax, ay, az, id);

		kvx_d[id + S * N] = ax;
		kvy_d[id + S * N] = ay;
		kvz_d[id + S * N] = az;
	}

}

//combine stage step and interpolation kernel
//all stages in the same kernel
template < const int Nperturbers, const int Ninterpolate, const int nn >
__global__ void stageStepAll_kernel(double *m_d, double *x_d, double *y_d, double *z_d, double *vx_d, double *vy_d, double *vz_d, double *xp_d, double *yp_d, double *zp_d, double *timep_d, double time0, double dt, int N, int useHelio, int GR){

	int idx = threadIdx.x;
	int id = blockIdx.x * blockDim.x + idx;

	//shared memory contains only the perturbers
	//the particle idx is stored in registers
	__shared__ double x_s[Nperturbers];
	__shared__ double y_s[Nperturbers];
	__shared__ double z_s[Nperturbers];
	__shared__ double m_s[Nperturbers];


	__shared__ double kx_s[nn * 6];
	__shared__ double ky_s[nn * 6];
	__shared__ double kz_s[nn * 6];

	__shared__ double kvx_s[nn * 6];
	__shared__ double kvy_s[nn * 6];
	__shared__ double kvz_s[nn * 6];
	
	//needs at least Nperturbers threads per block
	// *******************************************************
	//interpolate

	//do interpolation in every thread block
	//p is the particle index

	double Cx[Ninterpolate];
	double Cy[Ninterpolate];
	double Cz[Ninterpolate];

	double Dx[Ninterpolate];
	double Dy[Ninterpolate];
	double Dz[Ninterpolate];

	double tn[Ninterpolate];


	for(int S = 0; S < 6; ++S){

		double time = time0 + c_c[S] * dt / dayUnit;
		// **********************************3
		//interpolation
		if(idx < Nperturbers){

			for(int i = 0; i < Ninterpolate; ++i){
				Cx[i] = xp_d[idx * Ninterpolate + i];
				Cy[i] = yp_d[idx * Ninterpolate + i];
				Cz[i] = zp_d[idx * Ninterpolate + i];

				Dx[i] = Cx[i];		
				Dy[i] = Cy[i];		
				Dz[i] = Cz[i];		

				tn[i] = timep_d[idx * Ninterpolate + i];

		//printf("interpolate %d %d %.20g %.20g %.20g\n", idx, i, time, tn[i], Cx[i]);
			}

			//initialize with closest solution
			//Assume that the closest solution is in the middle

			int ii = Ninterpolate / 2 - 1;
			x_s[idx] = Cx[ii];
			y_s[idx] = Cy[ii];
			z_s[idx] = Cz[ii];
			m_s[idx] = m_d[idx]; 

			--ii;

			for(int j = 1; j < Ninterpolate; ++j){
		//printf("**** %d %d %g\n", j, ii, x_s[idx]);
				for(int i = 0; i < Ninterpolate - j; ++i){

					double dtn0 = tn[i] - time;
					double dtn1 = tn[i + j] - time;
					double dtn = tn[i] - tn[i + j];

					double dPx = (Cx[i + 1] - Dx[i]) / dtn;
					double dPy = (Cy[i + 1] - Dy[i]) / dtn;
					double dPz = (Cz[i + 1] - Dz[i]) / dtn;

					Dx[i] = dtn1 * dPx;
					Dy[i] = dtn1 * dPy;
					Dz[i] = dtn1 * dPz;

					Cx[i] = dtn0 * dPx;
					Cy[i] = dtn0 * dPy;
					Cz[i] = dtn0 * dPz;

			
		//printf("%d %d %g %g %g %g %.20g\n", i, i+j, tn[i], tn[i+j], P[j-1][i], P[j-1][i+1], P[j][i]);

				}

				if(2 * ii < Ninterpolate - j){
					x_s[idx] += Cx[ii + 1];
					y_s[idx] += Cy[ii + 1];
					z_s[idx] += Cz[ii + 1];
				}
				else{
					x_s[idx] += Dx[ii];
					y_s[idx] += Dy[ii];
					z_s[idx] += Dz[ii];
					--ii;
				}

			}

		}
		// end interpolation **********************************************

		__syncthreads();

		if(id >= Nperturbers && id < N){

			// ***********************
			//update
			double xi = x_d[id];
			double yi = y_d[id];
			double zi = z_d[id];
			double vxi = vx_d[id];
			double vyi = vy_d[id];
			double vzi = vz_d[id];
			double mi = m_d[id];

			for(int s = 0; s < S; ++s){
				double aa = a_c[S * 6 + s];
				xi  += dt * aa * kx_s[idx + s * nn];
				yi  += dt * aa * ky_s[idx + s * nn];
				zi  += dt * aa * kz_s[idx + s * nn];
				vxi += dt * aa * kvx_s[idx + s * nn];
				vyi += dt * aa * kvy_s[idx + s * nn];
				vzi += dt * aa * kvz_s[idx + s * nn];
			}
			// *****************************

			kx_s[idx + S * nn] = vxi;
			ky_s[idx + S * nn] = vyi;
			kz_s[idx + S * nn] = vzi;

			double ax = 0.0;
			double ay = 0.0;
			double az = 0.0;

			if(useHelio == 0){
				for(int j = Nperturbers-1; j >= 0; --j){
					accP_device(m_s[j], x_s[j], y_s[j], z_s[j], xi, yi, zi, ax, ay, az);
				}
			}
			else{
				for(int j = Nperturbers-1; j >= 1; --j){
					accP_device(m_s[j], x_s[j], y_s[j], z_s[j], xi, yi, zi, ax, ay, az);
	//printf("Nij %d %d %.20g %.20g %.20g\n", id, j, ax * dayUnit * dayUnit, ay * dayUnit * dayUnit, az * dayUnit * dayUnit);
				}
				accS_device(m_s[0] + mi, xi, yi, zi, ax, ay, az);

	//printf("N0 %d %.20g %.20g %.20g\n", id, ax * dayUnit * dayUnit, ay * dayUnit * dayUnit, az * dayUnit * dayUnit);
				for(int j = Nperturbers-1; j >= 1; --j){
					accP2_device(m_s[j], x_s[j], y_s[j], z_s[j], ax, ay, az);
	//printf("Npi %d %d %.20g %.20g %.20g %d\n", id, j, ax * dayUnit * dayUnit, ay * dayUnit * dayUnit, az * dayUnit * dayUnit, S);
				}
	//printf("Np %d %.20g %.20g %.20g %d\n", id, ax * dayUnit * dayUnit, ay * dayUnit * dayUnit, az * dayUnit * dayUnit, S);
			}


			if(GR == 2){
				acchGR2(xi, yi, zi, vxi, vyi, vzi, ax, ay, az);
			}

			//NonGrav(xt, yt, zt, vxt, vyt, vzt, ax, ay, az, A1, A2, A3, ALN, NK, NM, Nn, R0, id);
			//J2(m, xt, yt, zt, ax, ay, az, id);

			kvx_s[idx + S * nn] = ax;
			kvy_s[idx + S * nn] = ay;
			kvz_s[idx + S * nn] = az;
		}
		// end of stage
		__syncthreads();
	}

	if(id >= Nperturbers && id < N){
	//RKF45
		x_d[id] += dt * (b_c[0] * kx_s[idx + 0 * nn] + b_c[2] * kx_s[idx + 2 * nn] + b_c[3] * kx_s[idx + 3 * nn] + b_c[4] * kx_s[idx + 4 * nn] + b_c[5] * kx_s[idx + 5 * nn]);
		y_d[id] += dt * (b_c[0] * ky_s[idx + 0 * nn] + b_c[2] * ky_s[idx + 2 * nn] + b_c[3] * ky_s[idx + 3 * nn] + b_c[4] * ky_s[idx + 4 * nn] + b_c[5] * ky_s[idx + 5 * nn]);
		z_d[id] += dt * (b_c[0] * kz_s[idx + 0 * nn] + b_c[2] * kz_s[idx + 2 * nn] + b_c[3] * kz_s[idx + 3 * nn] + b_c[4] * kz_s[idx + 4 * nn] + b_c[5] * kz_s[idx + 5 * nn]);

		vx_d[id] += dt * (b_c[0] * kvx_s[idx + 0 * nn] + b_c[2] * kvx_s[idx + 2 * nn] + b_c[3] * kvx_s[idx + 3 * nn] + b_c[4] * kvx_s[idx + 4 * nn] + b_c[5] * kvx_s[idx + 5 * nn]);
		vy_d[id] += dt * (b_c[0] * kvy_s[idx + 0 * nn] + b_c[2] * kvy_s[idx + 2 * nn] + b_c[3] * kvy_s[idx + 3 * nn] + b_c[4] * kvy_s[idx + 4 * nn] + b_c[5] * kvy_s[idx + 5 * nn]);
		vz_d[id] += dt * (b_c[0] * kvz_s[idx + 0 * nn] + b_c[2] * kvz_s[idx + 2 * nn] + b_c[3] * kvz_s[idx + 3 * nn] + b_c[4] * kvz_s[idx + 4 * nn] + b_c[5] * kvz_s[idx + 5 * nn]);

	}

}

__host__ void update1(double *xt, double *yt, double *zt, double *vxt, double *vyt, double *vzt, double *x, double *y, double *z, double *vx, double *vy, double *vz, int i){

	xt[i] = x[i];
	yt[i] = y[i];
	zt[i] = z[i];
	vxt[i] = vx[i];
	vyt[i] = vy[i];
	vzt[i] = vz[i];
}


__host__ void update2(double *xt, double *yt, double *zt, double *vxt, double *vyt, double *vzt, double *x, double *y, double *z, double *vx, double *vy, double *vz, double *kx, double *ky, double *kz, double *kvx, double *kvy, double *kvz, int i, int N, double dt, int S, double *a){

	xt[i]  = x[i];
	yt[i]  = y[i];
	zt[i]  = z[i];
	vxt[i] = vx[i];
	vyt[i] = vy[i];
	vzt[i] = vz[i];

	for(int s = 0; s < S; ++s){
		xt[i]  += dt * a[S * 6 + s] * kx[i + s * N];
		yt[i]  += dt * a[S * 6 + s] * ky[i + s * N];
		zt[i]  += dt * a[S * 6 + s] * kz[i + s * N];
		vxt[i] += dt * a[S * 6 + s] * kvx[i + s * N];
		vyt[i] += dt * a[S * 6 + s] * kvy[i + s * N];
		vzt[i] += dt * a[S * 6 + s] * kvz[i + s * N];
	}
}


__host__ void update(double *x, double *y, double *z, double *vx, double *vy, double *vz, double *kx, double *ky, double *kz, double *kvx, double *kvy, double *kvz, int i, int N, double dt, double *b){
	//RKF45
	x[i] += dt * (b[0] * kx[i + 0 * N] + b[2] * kx[i + 2 * N] + b[3] * kx[i + 3 * N] + b[4] * kx[i + 4 * N] + b[5] * kx[i + 5 * N]);
	y[i] += dt * (b[0] * ky[i + 0 * N] + b[2] * ky[i + 2 * N] + b[3] * ky[i + 3 * N] + b[4] * ky[i + 4 * N] + b[5] * ky[i + 5 * N]);
	z[i] += dt * (b[0] * kz[i + 0 * N] + b[2] * kz[i + 2 * N] + b[3] * kz[i + 3 * N] + b[4] * kz[i + 4 * N] + b[5] * kz[i + 5 * N]);

	vx[i] += dt * (b[0] * kvx[i + 0 * N] + b[2] * kvx[i + 2 * N] + b[3] * kvx[i + 3 * N] + b[4] * kvx[i + 4 * N] + b[5] * kvx[i + 5 * N]);
	vy[i] += dt * (b[0] * kvy[i + 0 * N] + b[2] * kvy[i + 2 * N] + b[3] * kvy[i + 3 * N] + b[4] * kvy[i + 4 * N] + b[5] * kvy[i + 5 * N]);
	vz[i] += dt * (b[0] * kvz[i + 0 * N] + b[2] * kvz[i + 2 * N] + b[3] * kvz[i + 3 * N] + b[4] * kvz[i + 4 * N] + b[5] * kvz[i + 5 * N]);
}

__global__ void update_kernel(double *x, double *y, double *z, double *vx, double *vy, double *vz, double *kx, double *ky, double *kz, double *kvx, double *kvy, double *kvz, int N, int Nperturbers, double dt){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if(idx >= Nperturbers && idx < N){
	//RKF45
		x[idx] += dt * (b_c[0] * kx[idx + 0 * N] + b_c[2] * kx[idx + 2 * N] + b_c[3] * kx[idx + 3 * N] + b_c[4] * kx[idx + 4 * N] + b_c[5] * kx[idx + 5 * N]);
		y[idx] += dt * (b_c[0] * ky[idx + 0 * N] + b_c[2] * ky[idx + 2 * N] + b_c[3] * ky[idx + 3 * N] + b_c[4] * ky[idx + 4 * N] + b_c[5] * ky[idx + 5 * N]);
		z[idx] += dt * (b_c[0] * kz[idx + 0 * N] + b_c[2] * kz[idx + 2 * N] + b_c[3] * kz[idx + 3 * N] + b_c[4] * kz[idx + 4 * N] + b_c[5] * kz[idx + 5 * N]);

		vx[idx] += dt * (b_c[0] * kvx[idx + 0 * N] + b_c[2] * kvx[idx + 2 * N] + b_c[3] * kvx[idx + 3 * N] + b_c[4] * kvx[idx + 4 * N] + b_c[5] * kvx[idx + 5 * N]);
		vy[idx] += dt * (b_c[0] * kvy[idx + 0 * N] + b_c[2] * kvy[idx + 2 * N] + b_c[3] * kvy[idx + 3 * N] + b_c[4] * kvy[idx + 4 * N] + b_c[5] * kvy[idx + 5 * N]);
		vz[idx] += dt * (b_c[0] * kvz[idx + 0 * N] + b_c[2] * kvz[idx + 2 * N] + b_c[3] * kvz[idx + 3 * N] + b_c[4] * kvz[idx + 4 * N] + b_c[5] * kvz[idx + 5 * N]);
	}
}

int main(int argc, char*argv[]){

	//Number of planets
	const int NN = 27 + 1;//+ 8192 * 4; //28
	const int Nperturbers = 27;
	const int Ninterpolate = 10;	//number of interpolation points
	const double dtime = 1.0;     //interval between stored time steps

	int GR = 2;
	//2 Sitarski 1982, heliocentric coordinates

	int useGPU = 1;

	int useHelio = 1;
	int outHelio = 1;
	//1 print output in heliocentric coordinates
	//0 print output in barycentric  coordinates

	//long long int Nsteps = 40000;	
	//long long int outInterval = 10;
	//double dt = 0.1 * dayUnit;

	//long long int Nsteps = 400000;	
	long long int Nsteps = 40000;
	long long int outInterval = 100;
	double dt = 0.01 * dayUnit;

	for(int i = 1; i < argc; i += 2){

		if(strcmp(argv[i], "-Nsteps") == 0){
			Nsteps = atoll(argv[i + 1]);
		}
		else if(strcmp(argv[i], "-outInterval") == 0){
			outInterval = atoll(argv[i + 1]);
		}
		else if(strcmp(argv[i], "-dt") == 0){
			dt = atof(argv[i + 1]);
		}
	}

	
	int *id_h, *id_d;
	double *m_h, *m_d;
	double *x_h, *x_d;
	double *y_h, *y_d;
	double *z_h, *z_d;
	double *vx_h, *vx_d;
	double *vy_h, *vy_d;
	double *vz_h, *vz_d;

	//coordinates from data table
	double *timep_h, *timep_d;
	double *xp_h, *xp_d;
	double *yp_h, *yp_d;
	double *zp_h, *zp_d;


	//allocate data on host
	id_h = (int*)malloc(NN * sizeof(int));
	m_h = (double*)malloc(NN * sizeof(double));
	x_h = (double*)malloc(NN * sizeof(double));
	y_h = (double*)malloc(NN * sizeof(double));
	z_h = (double*)malloc(NN * sizeof(double));
	vx_h = (double*)malloc(NN * sizeof(double));
	vy_h = (double*)malloc(NN * sizeof(double));
	vz_h = (double*)malloc(NN * sizeof(double));

	timep_h = (double*)malloc(Nperturbers * Ninterpolate * sizeof(double));
	xp_h = (double*)malloc(Nperturbers * Ninterpolate * sizeof(double));
	yp_h = (double*)malloc(Nperturbers * Ninterpolate * sizeof(double));
	zp_h = (double*)malloc(Nperturbers * Ninterpolate * sizeof(double));

	//allocate data on the device
	hipMalloc((void **) &id_d, NN * sizeof(int));
	hipMalloc((void **) &m_d, NN * sizeof(double));
	hipMalloc((void **) &x_d, NN * sizeof(double));
	hipMalloc((void **) &y_d, NN * sizeof(double));
	hipMalloc((void **) &z_d, NN * sizeof(double));
	hipMalloc((void **) &vx_d, NN * sizeof(double));
	hipMalloc((void **) &vy_d, NN * sizeof(double));
	hipMalloc((void **) &vz_d, NN * sizeof(double));

	hipMalloc((void **) &timep_d, Nperturbers * Ninterpolate * sizeof(double));
	hipMalloc((void **) &xp_d, Nperturbers * Ninterpolate * sizeof(double));
	hipMalloc((void **) &yp_d, Nperturbers * Ninterpolate * sizeof(double));
	hipMalloc((void **) &zp_d, Nperturbers * Ninterpolate * sizeof(double));

	/*
	//non Gravitational constants
	double A1[NN];
	double A2[NN];
	double A3[NN];
	double ALN[NN];
	double NK[NN];
	double NM[NN];
	double Nn[NN];
	double R0[NN];

	for(int i = 0; i < NN; ++i){
		A1[i] = 0.0;
		A2[i] = 0.0;
		A3[i] = 0.0;
		ALN[i] = 0.0;
		NK[i] = 0.0;
		NM[i] = 0.0;
		Nn[i] = 0.0;
		R0[i] = 1.0;

	}
	*/

	FILE *outfile;
	char outfilename[160];	


	double time = 0.0;

	//Units are 1/(mass of object in solar masses)
	double pmass[] = {
		//The sun has to be at the first position
		1.000000000000000e0,      // Sun        (0)
		6023682.155592479e0,      // Mercury    (1)
		408523.7186582996e0,      // Venus      (2)
		332946.0488339480e0,      // Earth      (3)
		3098703.590290707e0,      // Mars       (4)
		1047.348625463337e0,      // Jupiter    (5)
		3497.901767786633e0,      // Saturn     (6)
		22902.98161308703e0,      // Uranus     (7)
		19412.25977597307e0,      // Neptune    (8)
		135836683.7686175e0,      // Pluto      (9)
		2112939391.8192508,                  // Ceres      (10)
		9531877787.0654011,                  // Pallas     (11)
		81799329362.428986,                  // Juno       (12)
		7676559929.1351004,                  // Vesta      (13)
		23944976514.662392,                  // Hygiea     (14)
		63251980219.354561,                  // Eunomia    (15)
		46649712166.264168,                  // Euphrosyne (16)
		119474172269.94408,                  // Europa     (17)
		56926698684.931702,                  // Davida     (18)
		56298080671.641434,                  // Interamnia (19)
		27068703.24120323e0,      // Moon       (20)

		86737410876.841156,		  // Psyche
		93034865412.812271,		  // Cybele
		114823090351.20033,		  // Thisbe
		116910898662.48077,		  // Doris
		128906361339.41116,		  // Patientia
		134548655333.38321,		  // Sylvia

		0.0			  // test particle

	};
	for(int i = 0; i < Nperturbers; ++i){
		m_h[i] = 1.0/pmass[i];
printf("m %d %.20g\n", i, m_h[i]);
	}

//m[Nperturbers] = 1.e-11; //ca mass of Flora

	int N = Nperturbers;

	//Sun
	FILE *infile;
	char infilename[160];

	sprintf(infilename, "initial.dat");
	infile = fopen(infilename, "r");
	//sun
	id_h[0] = 20;
	x_h[0] = 0.0;
	y_h[0] = 0.0;
	z_h[0] = 0.0;
	vx_h[0] = 0.0;
	vy_h[0] = 0.0;
	vz_h[0] = 0.0;

	for(int i = 1; i < NN; ++i){
		id_h[i] = -1;
		x_h[i] = 0.0;
		y_h[i] = 0.0;
		z_h[i] = 0.0;
		vx_h[i] = 0.0;
		vy_h[i] = 0.0;
		vz_h[i] = 0.0;
	}
	//read test particle
	for(int i = Nperturbers; i < NN; ++i){
		int er = 0;
		fscanf(infile, "%lf", &time);
		fscanf(infile, "%lf", &x_h[i]);
		fscanf(infile, "%lf", &y_h[i]);
		fscanf(infile, "%lf", &z_h[i]);
		fscanf(infile, "%lf", &vx_h[i]);
		fscanf(infile, "%lf", &vy_h[i]);
		er= fscanf(infile, "%lf", &vz_h[i]);
		//fscanf(infile, "%lf", &A1[i]);
		//fscanf(infile, "%lf", &A2[i]);
		//fscanf(infile, "%lf", &A3[i]);
		//fscanf(infile, "%lf", &ALN[i]);
		//fscanf(infile, "%lf", &NK[i]);
		//fscanf(infile, "%lf", &NM[i]);
		//fscanf(infile, "%lf", &Nn[i]);
		//er = fscanf(infile, "%lf", &R0[i]);
		if(er < 0) break;
		++N;
//printf("er %d %d %d %d %.20g %.20g %.20g\n", i, id_h[i], er, N, x_h[i], y_h[i], z_h[i]);
	}
	fclose(infile);
	double time0 = time;	//start time from simulation
	double time1 = time;	//time from table position

	for(int i = Nperturbers; i < N; ++i){
		vx_h[i] /= dayUnit;
		vy_h[i] /= dayUnit;
		vz_h[i] /= dayUnit;

		//A1[i] /= (dayUnit * dayUnit);
		//A2[i] /= (dayUnit * dayUnit);
		//A3[i] /= (dayUnit * dayUnit);
	}



	FILE *XVfile;
	if(useHelio == 1){
		XVfile = fopen("All_h.dat", "r");
	}
	else{
		XVfile = fopen("All_b.dat", "r");
	}


	// -----------------------------------------
	//Read table
	int countNodes = 0;
	for(int t = 0; t < 1000000; ++t){
		int er;
printf("CountNodes %d\n", countNodes);
		for(int i = 0; i < Nperturbers; ++i){
			double skip;
			double timepp;
			int id;
			er = fscanf(XVfile, "%lf %d", &timepp, &id);
			fscanf(XVfile, "%lf %lf %lf", &xp_h[id * Ninterpolate + countNodes], &yp_h[id * Ninterpolate + countNodes], &zp_h[id * Ninterpolate + countNodes]);
			//remove velocities read later
			fscanf(XVfile, "%lf %lf %lf", &skip, &skip, &skip);

			if(er < 0) break;
			timep_h[id * Ninterpolate + countNodes] = timepp;

//printf("read %.20g %d %.20g %.20g %.20g%d\n", timep_h[id * Ninterpolate + countNodes], id, xp_h[id * Ninterpolate + countNodes], yp_h[id * Ninterpolate + countNodes], zp_h[id * Ninterpolate + countNodes], id * Ninterpolate + countNodes);

			//vxp[id * Ninterpolate + countNodes] /= dayUnit;
			//vyp[id * Ninterpolate + countNodes] /= dayUnit;
			//vzp[id * Ninterpolate + countNodes] /= dayUnit;


			if(i == 0 && t == 0 && timep_h[id * Ninterpolate + countNodes] > time - (Ninterpolate/2 - 1) * dtime){
				printf("Error, time too small, not enough data before time\n");
				return 0;
			}
			if(i == Nperturbers - 1 && timep_h[id * Ninterpolate + countNodes] > time - Ninterpolate/2 * dtime){
				++countNodes;
			}
		}
		if(er < 0) break;
		if(countNodes >= Ninterpolate){
			break;
		}
	}
	if(countNodes < Ninterpolate){
		printf("Error, time too large, not enough data after time\n");
		return 0;
	}
	// ---------------------------------------

	

	// ---------------------------------------
	//interpolate on host
	for(int p = 0; p < Nperturbers; ++p){
		//interpolate(Ninterpolate, xp_h, yp_h, zp_h, timep_h, time, x_h, y_h, z_h, p);
		interpolate2(Ninterpolate, xp_h, yp_h, zp_h, timep_h, time, x_h, y_h, z_h, p);
	}
	// ---------------------------------------

	

	//first output
	double comx = 0.0;
	double comy = 0.0;
	double comz = 0.0;
	double vcomx = 0.0;
	double vcomy = 0.0;
	double vcomz = 0.0;
	double mtot = 0.0;	


	if(useHelio == 0 && outHelio == 1){
		//convert to heliocentric output

		comx = -x_h[0];
		comy = -y_h[0];
		comz = -z_h[0];
		vcomx = -vx_h[0];
		vcomy = -vy_h[0];
		vcomz = -vz_h[0];
	}
	
	if(outHelio == 1){
		sprintf(outfilename, "Outhelio10_%.12d.dat", 0);
	}
	else{
		sprintf(outfilename, "Outbary10_%.12d.dat", 0);
	}
	outfile = fopen(outfilename, "w");
	printf("%s\n", outfilename);
	for(int i = 0; i < N; ++i){
		fprintf(outfile, "%.10g %d %.40g %.40g %.40g %.40g %.40g %.40g %.40g\n", time, i, m_h[i], comx + x_h[i], comy + y_h[i], comz + z_h[i], vcomx + vx_h[i], vcomy + vy_h[i], vcomz + vz_h[i]);

	}
	fclose(outfile);


	//copy the data to the device
	if(useGPU == 1){
		hipMemcpy(m_d, m_h, NN * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(x_d, x_h, NN * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(y_d, y_h, NN * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(z_d, z_h, NN * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(vx_d, vx_h, NN * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(vy_d, vy_h, NN * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(vz_d, vz_h, NN * sizeof(double), hipMemcpyHostToDevice);

		hipMemcpy(timep_d, timep_h, Nperturbers * Ninterpolate * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(xp_d, xp_h, Nperturbers * Ninterpolate * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(yp_d, yp_h, Nperturbers * Ninterpolate * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(zp_d, zp_h, Nperturbers * Ninterpolate * sizeof(double), hipMemcpyHostToDevice);
	}
	

	//remove this later to shared memory
	double *xt_h, *xt_d;
	double *yt_h, *yt_d;
	double *zt_h, *zt_d;
	double *vxt_h, *vxt_d;
	double *vyt_h, *vyt_d;
	double *vzt_h, *vzt_d;
	
	double *kx_h, *kx_d;
	double *ky_h, *ky_d;
	double *kz_h, *kz_d;
	double *kvx_h, *kvx_d;
	double *kvy_h, *kvy_d;
	double *kvz_h, *kvz_d;

	xt_h = (double*)malloc(NN * sizeof(double));
	yt_h = (double*)malloc(NN * sizeof(double));
	zt_h = (double*)malloc(NN * sizeof(double));
	vxt_h = (double*)malloc(NN * sizeof(double));
	vyt_h = (double*)malloc(NN * sizeof(double));
	vzt_h = (double*)malloc(NN * sizeof(double));

	kx_h = (double*)malloc(NN * 6 * sizeof(double));
	ky_h = (double*)malloc(NN * 6 * sizeof(double));
	kz_h = (double*)malloc(NN * 6 * sizeof(double));
	kvx_h = (double*)malloc(NN * 6 * sizeof(double));
	kvy_h = (double*)malloc(NN * 6 * sizeof(double));
	kvz_h = (double*)malloc(NN * 6 * sizeof(double));

	hipMalloc((void **) &xt_d, NN * sizeof(double));
	hipMalloc((void **) &yt_d, NN * sizeof(double));
	hipMalloc((void **) &zt_d, NN * sizeof(double));
	hipMalloc((void **) &vxt_d, NN * sizeof(double));
	hipMalloc((void **) &vyt_d, NN * sizeof(double));
	hipMalloc((void **) &vzt_d, NN * sizeof(double));

	hipMalloc((void **) &kx_d, NN * 6 * sizeof(double));
	hipMalloc((void **) &ky_d, NN * 6 * sizeof(double));
	hipMalloc((void **) &kz_d, NN * 6 * sizeof(double));
	hipMalloc((void **) &kvx_d, NN * 6 * sizeof(double));
	hipMalloc((void **) &kvy_d, NN * 6 * sizeof(double));
	hipMalloc((void **) &kvz_d, NN * 6 * sizeof(double));

	//double errorkx[N];
	//double errorky[N];
	//double errorkz[N];
	//double errorkvx[N];
	//double errorkvy[N];
	//double errorkvz[N];

	
	double *a_h;
	a_h = (double*)malloc(6 * 6 * sizeof(double));

	a_h[1 * 6 + 0] = 1.0/4.0;	//21

	a_h[2 * 6 + 0] = 3.0/32.0;	//31
	a_h[2 * 6 + 1] = 9.0/32.0;	//32

	a_h[3 * 6 + 0] = 1932.0/2197.0;	//41
	a_h[3 * 6 + 1] = -7200.0/2197.0;//42
	a_h[3 * 6 + 2] = 7296.0/2197.0;	//43

	a_h[4 * 6 + 0] = 439.0/216.0;	//51
	a_h[4 * 6 + 1] = -8.0;		//52
	a_h[4 * 6 + 2] = 3680.0/513.0;	//53
	a_h[4 * 6 + 3] = -845.0/4104.0;	//54

	a_h[5 * 6 + 0] = -8.0/27.0;	//61
	a_h[5 * 6 + 1] = 2.0;		//62
	a_h[5 * 6 + 2] = -3544/2565.0;	//63
	a_h[5 * 6 + 3] = 1859.0/4104.0;	//64
	a_h[5 * 6 + 4] = -11.0/40.0;	//65

	double *b_h, *bb_h;
	b_h = (double*)malloc(6 * sizeof(double));
	bb_h = (double*)malloc(6 * sizeof(double));

	b_h[0] = 25.0/216.0;
	b_h[1] = 0.0;
	b_h[2] = 1408.0/2565.0;
	b_h[3] = 2197.0/4104.0;
	b_h[4] = -1.0/5.0;
	b_h[5] = 0.0;

	bb_h[0] = 16.0/135.0;
	bb_h[1] = 0.0;
	bb_h[2] = 6656.0/12825.0;
	bb_h[3] = 28561.0/56430.0;
	bb_h[4] = -9.0/50.0;
	bb_h[5] = 2.0/55.0;

	double *c_h;
	c_h = (double*)malloc(6 * sizeof(double));

	c_h[0] = 0.0;
	c_h[1] = 0.25;
	c_h[2] = 3.0 / 8.0;
	c_h[3] = 12.0 / 13.0;
	c_h[4] = 1.0;
	c_h[5] = 0.5;

	if(useGPU == 1){
		hipMemcpyToSymbol(HIP_SYMBOL(a_c), a_h, 6 * 6 * sizeof(double), 0, hipMemcpyHostToDevice);
		hipMemcpyToSymbol(HIP_SYMBOL(b_c), b_h, 6 * sizeof(double), 0, hipMemcpyHostToDevice);
		hipMemcpyToSymbol(HIP_SYMBOL(c_c), c_h, 6 * sizeof(double), 0, hipMemcpyHostToDevice);
	}


	int S;
	for(long long int t = 1; t <= Nsteps; ++t){

			
		//stage 1
		S = 0;
		if(useGPU == 0){
			for(int i = 0; i < N; ++i){
				update1(xt_h, yt_h, zt_h, vxt_h, vyt_h, vzt_h, x_h, y_h, z_h, vx_h, vy_h, vz_h, i);
			}
			for(int i = Nperturbers; i < N; ++i){
				stageStep(m_h, xt_h, yt_h, zt_h, vxt_h, vyt_h, vzt_h, kx_h, ky_h, kz_h, kvx_h, kvy_h, kvz_h, S, i, Nperturbers, N, useHelio, GR);
			}
		}
		else{
			//stageStep_kernel < Nperturbers > <<< (NN + 127) / 128, 128 >>> (m_d, x_d, y_d, z_d, vx_d, vy_d, vz_d, xt_d, yt_d, zt_d, vxt_d, vyt_d, vzt_d, kx_d, ky_d, kz_d, kvx_d, kvy_d, kvz_d, dt, S, Nperturbers, N, useHelio, GR);
			stageStep1_kernel < Nperturbers, Ninterpolate > <<< (NN + 127) / 128, 128 >>> (m_d, x_d, y_d, z_d, vx_d, vy_d, vz_d, xp_d, yp_d, zp_d, timep_d, time + c_h[S] * dt / dayUnit, kx_d, ky_d, kz_d, kvx_d, kvy_d, kvz_d, dt, S, Nperturbers, N, useHelio, GR);
		}	
				
		//stage 2 - 6
		for(int S = 1; S < 6; ++S){
			if(useGPU == 0){
				for(int p = 0; p < Nperturbers; ++p){
					//interpolate(Ninterpolate, xp_h, yp_h, zp_h, timep_h, time + c_h[S] * dt / dayUnit, xt_h, yt_h, zt_h, p);
					interpolate2(Ninterpolate, xp_h, yp_h, zp_h, timep_h, time + c_h[S] * dt / dayUnit, xt_h, yt_h, zt_h, p);
				}
				for(int i = Nperturbers; i < N; ++i){
					update2(xt_h, yt_h, zt_h, vxt_h, vyt_h, vzt_h, x_h, y_h, z_h, vx_h, vy_h, vz_h, kx_h, ky_h, kz_h, kvx_h, kvy_h, kvz_h, i, N, dt, S, a_h);	//a21
				}
				for(int i = Nperturbers; i < N; ++i){
					stageStep(m_h, xt_h, yt_h, zt_h, vxt_h, vyt_h, vzt_h, kx_h, ky_h, kz_h, kvx_h, kvy_h, kvz_h, S, i, Nperturbers, N, useHelio, GR);
				}
			}
			else{
				//interpolate_kernel < Ninterpolate > <<< Nperturbers, Ninterpolate >>> (Nperturbers, xp_d, yp_d, zp_d, timep_d, time + c_h[S] * dt / dayUnit, xt_d, yt_d, zt_d);
				//stageStep_kernel < Nperturbers > <<< (NN + 127) / 128, 128 >>> (m_d, x_d, y_d, z_d, vx_d, vy_d, vz_d, xt_d, yt_d, zt_d, vxt_d, vyt_d, vzt_d, kx_d, ky_d, kz_d, kvx_d, kvy_d, kvz_d, dt, S, Nperturbers, N, useHelio, GR);
				stageStep1_kernel < Nperturbers, Ninterpolate > <<< (NN + 127) / 128, 128 >>> (m_d, x_d, y_d, z_d, vx_d, vy_d, vz_d, xp_d, yp_d, zp_d, timep_d, time + c_h[S] * dt / dayUnit, kx_d, ky_d, kz_d, kvx_d, kvy_d, kvz_d, dt, S, Nperturbers, N, useHelio, GR);

			}
		}
//		stageStepAll_kernel < Nperturbers, Ninterpolate, 128 > <<< (NN + 127) / 128, 128 >>> (m_d, x_d, y_d, z_d, vx_d, vy_d, vz_d, xp_d, yp_d, zp_d, timep_d, time, dt, N, useHelio, GR);
	
		double sc = 1.0e-15;

		/*	
		//error estimation
		for(int i = 0; i < N; ++i){
			errorkx[i] = ((b1 - bb1) * kx[i][0] + (b3 - bb3) * kx[i][2] + (b4 - bb4) * kx[i][3] + (b5 - bb5) * kx[i][4] + (b6 - bb6) * kx[i][5]) / sc;
			errorky[i] = ((b1 - bb1) * ky[i][0] + (b3 - bb3) * ky[i][2] + (b4 - bb4) * ky[i][3] + (b5 - bb5) * ky[i][4] + (b6 - bb6) * ky[i][5]) / sc;
			errorkz[i] = ((b1 - bb1) * kz[i][0] + (b3 - bb3) * kz[i][2] + (b4 - bb4) * kz[i][3] + (b5 - bb5) * kz[i][4] + (b6 - bb6) * kz[i][5]) / sc;
			errorkvx[i] = ((b1 - bb1) * kvx[i][0] + (b3 - bb3) * kvx[i][2] + (b4 - bb4) * kvx[i][3] + (b5 - bb5) * kvx[i][4] + (b6 - bb6) * kvx[i][5]) / sc;
			errorkvy[i] = ((b1 - bb1) * kvy[i][0] + (b3 - bb3) * kvy[i][2] + (b4 - bb4) * kvy[i][3] + (b5 - bb5) * kvy[i][4] + (b6 - bb6) * kvy[i][5]) / sc;
			errorkvz[i] = ((b1 - bb1) * kvz[i][0] + (b3 - bb3) * kvz[i][2] + (b4 - bb4) * kvz[i][3] + (b5 - bb5) * kvz[i][4] + (b6 - bb6) * kvz[i][5]) / sc;
		}


		double error = 0.0;
		for(int i = 0; i < N; ++i){
			error += errorkx[i] * errorkx[i];
			error += errorky[i] * errorky[i];
			error += errorkz[i] * errorkz[i];
			error += errorkvx[i] * errorkvx[i];
			error += errorkvy[i] * errorkvy[i];
			error += errorkvz[i] * errorkvz[i];
		}
		

		
		double errmax = 0.0;
		for(int i = 0; i < N; ++i){
			ermax = fmax(ermax, fabs(errorkx[i]));
			ermax = fmax(ermax, fabs(errorky[i]));
			ermax = fmax(ermax, fabs(errorkz[i]));
			ermax = fmax(ermax, fabs(errorkvx[i]));
			ermax = fmax(ermax, fabs(errorkvy[i]));
			ermax = fmax(ermax, fabs(errorkvz[i]));
		}
		

		
		double ee = 1.0/5.0;	

		double s = pow( 1.0  / error, ee);
		*/

		//printf("%g %g\n", dt, s);			

		
		if(useGPU == 0){
			for(int i = Nperturbers; i < N; ++i){
				update(x_h, y_h, z_h, vx_h, vy_h, vz_h, kx_h, ky_h, kz_h, kvx_h, kvy_h, kvz_h, i, N, dt, b_h);	
			}
		}
		else{
			update_kernel <<< (NN + 127) / 128, 128 >>> (x_d, y_d, z_d, vx_d, vy_d, vz_d, kx_d, ky_d, kz_d, kvx_d, kvy_d, kvz_d, N, Nperturbers, dt);	
		}
		
		time = time0 + t * dt / dayUnit;

		//update table
		if(time - time1 >= dtime){
			int countNodes = Ninterpolate - 1;
			int er;
			for(int j = 0; j < Ninterpolate - 1; ++j){
				for(int i = 0; i < Nperturbers; ++i){
					xp_h[i * Ninterpolate + j] = xp_h[i * Ninterpolate + j + 1];
					yp_h[i * Ninterpolate + j] = yp_h[i * Ninterpolate + j + 1];
					zp_h[i * Ninterpolate + j] = zp_h[i * Ninterpolate + j + 1];
					timep_h[i * Ninterpolate + j] = timep_h[i * Ninterpolate + j + 1];
				}
			}

//printf("CountNodes %d\n", countNodes);
			for(int i = 0; i < Nperturbers; ++i){
				double skip;
				double timepp;
				int id;
				er = fscanf(XVfile, "%lf %d", &timepp, &id);
				fscanf(XVfile, "%lf %lf %lf", &xp_h[id * Ninterpolate + countNodes], &yp_h[id * Ninterpolate + countNodes], &zp_h[id * Ninterpolate + countNodes]);
				//remove velocities read later
				fscanf(XVfile, "%lf %lf %lf", &skip, &skip, &skip);

				//vxp[id * Ninterpolate + countNodes] /= dayUnit;
				//vyp[id * Ninterpolate + countNodes] /= dayUnit;
				//vzp[id * Ninterpolate + countNodes] /= dayUnit;

				if(er < 0) break;
				timep_h[id * Ninterpolate + countNodes] = timepp;
	
//printf("%.20g %d %.20g %.20g %.20g %d\n", timep_h[id * Ninterpolate + countNodes], id, xp_h[id * Ninterpolate + countNodes], yp_h[id * Ninterpolate + countNodes], zp_h[id * Ninterpolate + countNodes], id * Ninterpolate + countNodes);

			}
			if(er < 0){
				printf("Error, time too large, not enough data after time\n");
				return 0;
			}
			time1 = time;
			if(useGPU == 1){
				hipMemcpy(timep_d, timep_h, Nperturbers * Ninterpolate * sizeof(double), hipMemcpyHostToDevice);
				hipMemcpy(xp_d, xp_h, Nperturbers * Ninterpolate * sizeof(double), hipMemcpyHostToDevice);
				hipMemcpy(yp_d, yp_h, Nperturbers * Ninterpolate * sizeof(double), hipMemcpyHostToDevice);
				hipMemcpy(zp_d, zp_h, Nperturbers * Ninterpolate * sizeof(double), hipMemcpyHostToDevice);
			}
		}
		
		// ---------------------------------------
		//interpolate on host
		if(useGPU == 0){
			for(int p = 0; p < Nperturbers; ++p){
				//interpolate(Ninterpolate, xp_h, yp_h, zp_h, timep_h, time, x_h, y_h, z_h, p);
				interpolate2(Ninterpolate, xp_h, yp_h, zp_h, timep_h, time, x_h, y_h, z_h, p);
			}
		}
		else{
			interpolate_kernel < Ninterpolate > <<< Nperturbers, Ninterpolate >>> (Nperturbers, xp_d, yp_d, zp_d, timep_d, time, x_d, y_d, z_d);
		}
		// ---------------------------------------

		
		
		if(t % outInterval == 0){
			if(useGPU == 1){
				hipMemcpy(x_h, x_d, NN * sizeof(double), hipMemcpyDeviceToHost);
				hipMemcpy(y_h, y_d, NN * sizeof(double), hipMemcpyDeviceToHost);
				hipMemcpy(z_h, z_d, NN * sizeof(double), hipMemcpyDeviceToHost);
				hipMemcpy(vx_h, vx_d, NN * sizeof(double), hipMemcpyDeviceToHost);
				hipMemcpy(vy_h, vy_d, NN * sizeof(double), hipMemcpyDeviceToHost);
				hipMemcpy(vz_h, vz_d, NN * sizeof(double), hipMemcpyDeviceToHost);
			}
			
			if(outHelio == 1){
				sprintf(outfilename, "Outhelio10_%.12lld.dat", t);
			}
			else{
				sprintf(outfilename, "Outbary10_%.12lld.dat", t);
			}
			outfile = fopen(outfilename, "w");
//			printf("%s\n", outfilename);

			comx = 0.0;
			comy = 0.0;
			comz = 0.0;
			vcomx = 0.0;
			vcomy = 0.0;
			vcomz = 0.0;
			mtot = 0.0;

			if(useHelio == 0 && outHelio == 1){
				//convert to heliocentric output
				comx = -x_h[0];
				comy = -y_h[0];
				comz = -z_h[0];
				vcomx = -vx_h[0];
				vcomy = -vy_h[0];
				vcomz = -vz_h[0];
			}
			
			for(int i = 0; i < N; ++i){
				fprintf(outfile, "%.10g %d %.40g %.40g %.40g %.40g %.40g %.40g %.40g\n", time, i, m_h[i], comx + x_h[i], comy + y_h[i], comz + z_h[i], vcomx + vx_h[i], vcomy + vy_h[i], vcomz + vz_h[i]);
				
			}
			fclose(outfile);
		}
		
	
		/*
		if( s < 1.0){
			dt *= 0.5;
		}
		if( s > 3.0){
			dt *= 2.0;
		}
		*/
	}	// end of time step loop
	fclose(XVfile);
	hipDeviceSynchronize();
	
}
	
