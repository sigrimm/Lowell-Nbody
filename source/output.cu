#include "hip/hip_runtime.h"
#include "Host.h"
__host__ void Host::output(unsigned long long int t, double time){

	printf("Output %.20g %llu\n", time, t);


	FILE *outfile;
	char outfilename[160];
	
	if(outHelio == 1){
		if(outBinary == 0){	
			sprintf(outfilename, "Outhelio_%.12llu.dat", t);
		}
		else{
			sprintf(outfilename, "Outhelio.bin");
		}
	}
	else{
		if(outBinary == 0){
			sprintf(outfilename, "Outbary_%.12llu.dat", t);
		}
		else{
			sprintf(outfilename, "Outbary.bin");
		}
	}
	if(outBinary == 0){
		outfile = fopen(outfilename, "w");
	}
	else{
		if(t == 0){
			outfile = fopen(outfilename, "wb");
		}
		else{
			outfile = fopen(outfilename, "ab");
		}
	}


	if(useGPU > 0 && t > 0){
		hipMemcpy(snew_h, snew_d, N * sizeof(double2), hipMemcpyDeviceToHost);
		hipMemcpy(dtmin_h, dtmin_d, N * sizeof(double), hipMemcpyDeviceToHost);
		hipMemcpy(x0_h, x0_d, N * sizeof(double), hipMemcpyDeviceToHost);
		hipMemcpy(y0_h, y0_d, N * sizeof(double), hipMemcpyDeviceToHost);
		hipMemcpy(z0_h, z0_d, N * sizeof(double), hipMemcpyDeviceToHost);
		hipMemcpy(vx0_h, vx0_d, N * sizeof(double), hipMemcpyDeviceToHost);
		hipMemcpy(vy0_h, vy0_d, N * sizeof(double), hipMemcpyDeviceToHost);
		hipMemcpy(vz0_h, vz0_d, N * sizeof(double), hipMemcpyDeviceToHost);

		hipMemcpy(xTable_h, xTable_d, Nperturbers * RKFn * sizeof(double), hipMemcpyDeviceToHost);
		hipMemcpy(yTable_h, yTable_d, Nperturbers * RKFn * sizeof(double), hipMemcpyDeviceToHost);
		hipMemcpy(zTable_h, zTable_d, Nperturbers * RKFn * sizeof(double), hipMemcpyDeviceToHost);
	}



//	printf("%s\n", outfilename);

	double comx = 0.0;
	double comy = 0.0;
	double comz = 0.0;
	double vcomx = 0.0;
	double vcomy = 0.0;
	double vcomz = 0.0;

	if(useHelio == 0 && outHelio == 1){
		//convert to heliocentric output
		comx = -x_h[0];
		comy = -y_h[0];
		comz = -z_h[0];
		vcomx = -vx_h[0];
		vcomy = -vy_h[0];
		vcomz = -vz_h[0];
	}
	
	if(outBinary == 0){
		//for(int p = 0; p < Nperturbers; ++p){
		//	int ii = p * RKFn + 12;	
		//	fprintf(outfile, "%.10g %llu %.40g %.40g %.40g %.40g %.40g %.40g %.40g %g\n", time, id_h[p], m_h[p], xTable_h[ii], yTable_h[ii], zTable_h[ii], 0.0, 0.0, 0.0, 0.0);
		//}
		for(int i = Nperturbers; i < N; ++i){
			//printf("%.10g %llu %.40g %.40g %.40g %.40g %.40g %.40g %.40g %g\n", time, id0_h[i], m0_h[i], comx + x0_h[i], comy + y0_h[i], comz + z0_h[i], (vcomx + vx0_h[i]) * dayUnit, (vcomy + vy0_h[i]) * dayUnit, (vcomz + vz0_h[i]) * dayUnit, dtmin_h[i]);
			fprintf(outfile, "%.10g %llu %.40g %.40g %.40g %.40g %.40g %.40g %.40g %g\n", time, id0_h[i], m0_h[i], comx + x0_h[i], comy + y0_h[i], comz + z0_h[i], (vcomx + vx0_h[i]) * dayUnit, (vcomy + vy0_h[i]) * dayUnit, (vcomz + vz0_h[i]) * dayUnit, dtmin_h[i]);
		}
	}
	else{
		for(int i = Nperturbers; i < N; ++i){

			//unsigned long long int id = id_h[i];
			unsigned long long int id = __builtin_bswap64 (id_h[i]);
			double xx = comx + x0_h[i];
			double yy = comy + y0_h[i];
			double zz = comz + z0_h[i];
			double vxx = (vcomx + vx0_h[i]) * dayUnit;
			double vyy = (vcomy + vy0_h[i]) * dayUnit;
			double vzz = (vcomz + vz0_h[i]) * dayUnit;

			fwrite(&id, sizeof(unsigned long long int), 1, outfile);
			fwrite(&time, sizeof(double), 1, outfile);
			fwrite(&xx, sizeof(double), 1, outfile);
			fwrite(&yy, sizeof(double), 1, outfile);
			fwrite(&zz, sizeof(double), 1, outfile);
			fwrite(&vxx, sizeof(double), 1, outfile);
			fwrite(&vyy, sizeof(double), 1, outfile);
			fwrite(&vzz, sizeof(double), 1, outfile);
			fwrite(&dtmin_h[i], sizeof(double), 1, outfile);

			//printf("%llu %g %g %g %g %g %g %g\n", id, time, xx, yy, zz, vxx, vyy, vzz);
		}
		
	}
	fclose(outfile);
}

