#include "hip/hip_runtime.h"
#include "Host.h"

//initialize variables
__host__ Host::Host(){

	timep0 = 0.0;

	comet = 0;

	useGR = 2;
	useJ2 = 1;
	useNonGrav = 1;

	useGPU = 1;
	useAdaptiveTimeSteps = 1;
	useIndividualTimeSteps = 0;

	useFIFO = 2;
	InVersion= 0.0;
	DoPreIntegration = 0;	

	useHelio = 1;
	outHelio = 1;

	outBinary = 0;

	Nsteps = 1e9;
	outInterval = 1;


	time0 = 0.0;
	time1 = 0.0;
	outStart = 0.0;
	time = 0.0;

	dti = 0.05;
	dt = dti * dayUnit;

	dts = 0.01;
	outI = 1llu;

	N = Nperturbers;
	NMax = 10000000;

	nRuns = 1;
	dtiMin = new double[10];
	runsN = new int[10];		//assume maximally 10 levels
	runsdt = new double[10];
	for(int i = 0; i < 10; ++i){
		runsN[i] = Nperturbers;
		runsdt[i] = dti;
		dtiMin[i] = 0.01;
	}

	outfilename = new char[160];
	dtfilename = new char[160];
	sprintf(dtfilename, "timesteps.dat");

	//Erase Outbinary file
	if(outBinary == 1){
		if(outHelio == 1){
			sprintf(outfilename, "Outhelio.bin");
		}
		else{
			sprintf(outfilename, "Outbary.bin");
		}
		outfile = fopen(outfilename, "wb");
		fclose(outfile);
	}

}

__host__ int Host::readparam(int argc, char*argv[]){

	FILE *paramfile;
	paramfile = fopen("param.dat", "r");


        char sp[160];
        int er;

	for(int j = 0; j < 1000; ++j){ //loop around all lines in the param.dat file
		int c;
		for(int i = 0; i < 50; ++i){
			c = fgetc(paramfile);
			if(c == EOF) break;
			sp[i] = char(c);
			if(c == '=' || c == ':'){
				sp[i + 1] = '\0';
				break;
			}
		}
		if(c == EOF) break;

		if(strcmp(sp, "Number of time step levels =") == 0){
			er = fscanf (paramfile, "%d", &nRuns);
			if(er <= 0){
				printf("Error: Number of time step levels is not valid!\n");
				return 0;
			}
			fgets(sp, 3, paramfile);
		}
		else if(strcmp(sp, "dt =") == 0){
			for(int t = 0; t < nRuns; ++t){
				er = fscanf (paramfile, "%lf", &runsdt[t]);
				if(er <= 0){
					printf("Error: dtMin value is not valid!\n");
					return 0;
				}
			}
			fgets(sp, 3, paramfile);
		}
		else if(strcmp(sp, "dtMin =") == 0){
			for(int t = 0; t < nRuns; ++t){
				er = fscanf (paramfile, "%lf", &dtiMin[t]);
				if(er <= 0){
					printf("Error: dtMin value is not valid!\n");
					return 0;
				}
			}
			fgets(sp, 3, paramfile);
		}
	}




	fclose(paramfile);
	return 1;
}


__host__ void Host::Alloc1(){
	//allocate data on host
	id_h = (unsigned long long int*)malloc(N * sizeof(unsigned long long int));
	index_h = (unsigned int*)malloc(N * sizeof(unsigned int));
	m_h = (double*)malloc(N * sizeof(double));
	x_h = (double*)malloc(N * sizeof(double));
	y_h = (double*)malloc(N * sizeof(double));
	z_h = (double*)malloc(N * sizeof(double));
	vx_h = (double*)malloc(N * sizeof(double));
	vy_h = (double*)malloc(N * sizeof(double));
	vz_h = (double*)malloc(N * sizeof(double));
	A1_h = (double*)malloc(N * sizeof(double));
	A2_h = (double*)malloc(N * sizeof(double));
	A3_h = (double*)malloc(N * sizeof(double));
	jd_init_h = (double*)malloc(N * sizeof(double));

	timep_h = (double*)malloc(Nperturbers * NTable * sizeof(double));
	xp_h = (double*)malloc(Nperturbers * NTable * sizeof(double));
	yp_h = (double*)malloc(Nperturbers * NTable * sizeof(double));
	zp_h = (double*)malloc(Nperturbers * NTable * sizeof(double));

	x0_h = (double*)malloc(N * sizeof(double));
	y0_h = (double*)malloc(N * sizeof(double));
	z0_h = (double*)malloc(N * sizeof(double));
	vx0_h = (double*)malloc(N * sizeof(double));
	vy0_h = (double*)malloc(N * sizeof(double));
	vz0_h = (double*)malloc(N * sizeof(double));
	A10_h = (double*)malloc(N * sizeof(double));
	A20_h = (double*)malloc(N * sizeof(double));
	A30_h = (double*)malloc(N * sizeof(double));
	m0_h = (double*)malloc(N * sizeof(double));
	id0_h = (unsigned long long int*)malloc(N * sizeof(unsigned long long int));
	index0_h = (unsigned int*)malloc(N * sizeof(unsigned int));

	xb_h = (double*)malloc(N * sizeof(double));
	yb_h = (double*)malloc(N * sizeof(double));
	zb_h = (double*)malloc(N * sizeof(double));
	vxb_h = (double*)malloc(N * sizeof(double));
	vyb_h = (double*)malloc(N * sizeof(double));
	vzb_h = (double*)malloc(N * sizeof(double));
	A1b_h = (double*)malloc(N * sizeof(double));
	A2b_h = (double*)malloc(N * sizeof(double));
	A3b_h = (double*)malloc(N * sizeof(double));
	mb_h = (double*)malloc(N * sizeof(double));
	idb_h = (unsigned long long int*)malloc(N * sizeof(unsigned long long int));

	dx_h = (double*)malloc(N * sizeof(double));
	dy_h = (double*)malloc(N * sizeof(double));
	dz_h = (double*)malloc(N * sizeof(double));
	dvx_h = (double*)malloc(N * sizeof(double));
	dvy_h = (double*)malloc(N * sizeof(double));
	dvz_h = (double*)malloc(N * sizeof(double));

	if(useGPU > 0){
		//allocate data on the device
		hipMalloc((void **) &id_d, N * sizeof(unsigned long long int));
		hipMalloc((void **) &index_d, N * sizeof(unsigned int));
		hipMalloc((void **) &m_d, N * sizeof(double));
		hipMalloc((void **) &x_d, N * sizeof(double));
		hipMalloc((void **) &y_d, N * sizeof(double));
		hipMalloc((void **) &z_d, N * sizeof(double));
		hipMalloc((void **) &vx_d, N * sizeof(double));
		hipMalloc((void **) &vy_d, N * sizeof(double));
		hipMalloc((void **) &vz_d, N * sizeof(double));
		hipMalloc((void **) &A1_d, N * sizeof(double));
		hipMalloc((void **) &A2_d, N * sizeof(double));
		hipMalloc((void **) &A3_d, N * sizeof(double));
		hipMalloc((void **) &jd_init_d, N * sizeof(double));

		hipMalloc((void **) &id0_d, N * sizeof(unsigned long long int));
		hipMalloc((void **) &index0_d, N * sizeof(unsigned int));
		hipMalloc((void **) &m0_d, N * sizeof(double));
		hipMalloc((void **) &x0_d, N * sizeof(double));
		hipMalloc((void **) &y0_d, N * sizeof(double));
		hipMalloc((void **) &z0_d, N * sizeof(double));
		hipMalloc((void **) &vx0_d, N * sizeof(double));
		hipMalloc((void **) &vy0_d, N * sizeof(double));
		hipMalloc((void **) &vz0_d, N * sizeof(double));
		hipMalloc((void **) &A10_d, N * sizeof(double));
		hipMalloc((void **) &A20_d, N * sizeof(double));
		hipMalloc((void **) &A30_d, N * sizeof(double));

		hipMalloc((void **) &timep_d, Nperturbers * NTable * sizeof(double));
		hipMalloc((void **) &xp_d, Nperturbers * NTable * sizeof(double));
		hipMalloc((void **) &yp_d, Nperturbers * NTable * sizeof(double));
		hipMalloc((void **) &zp_d, Nperturbers * NTable * sizeof(double));

		hipMalloc((void **) &dx_d, N * sizeof(double));
		hipMalloc((void **) &dy_d, N * sizeof(double));
		hipMalloc((void **) &dz_d, N * sizeof(double));
		hipMalloc((void **) &dvx_d, N * sizeof(double));
		hipMalloc((void **) &dvy_d, N * sizeof(double));
		hipMalloc((void **) &dvz_d, N * sizeof(double));
	}

	//perturbers data
	if(useGPU == 0){
		XYdata_h = (double*)malloc(Nperturbers * NTable * 4 * sizeof(double));
		readBufferA_h = (double*)malloc(Nperturbers * 4 * sizeof(double));
		readBufferB_h = (double*)malloc(Nperturbers * 4 * sizeof(double));
	}
	else{
		//allocate data for interleaved data transfer
		hipHostAlloc((void **) &readBufferA_h, Nperturbers * 4 * sizeof(double), hipHostMallocDefault);
		hipHostAlloc((void **) &readBufferB_h, Nperturbers * 4 * sizeof(double), hipHostMallocDefault);
		hipMalloc((void **) &XYdata_d, Nperturbers * NTable * 4 * sizeof(double));
	}

}

__host__ void Host::initialize1(){
	for(int i = 0; i < N; ++i){
		A1_h[i] = 0.0;
		A2_h[i] = 0.0;
		A3_h[i] = 0.0;
		jd_init_h[i] = 0.0;
	}

	//Sun
	id_h[0] = 10;
	index_h[0] = 0;
	m_h[0] = 1.0;
	x_h[0] = 0.0;
	y_h[0] = 0.0;
	z_h[0] = 0.0;
	vx_h[0] = 0.0;
	vy_h[0] = 0.0;
	vz_h[0] = 0.0;

	for(int i = 1; i < N; ++i){
		id_h[i] = Nperturbers + i;
		index_h[i] = i;
		m_h[i] = 0.0;
		x_h[i] = 0.0;
		y_h[i] = 0.0;
		z_h[i] = 0.0;
		vx_h[i] = 0.0;
		vy_h[i] = 0.0;
		vz_h[i] = 0.0;
	}
}

__host__ void Host::copy1(){

	hipMemcpy(m_d, m_h, N * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(id_d, id_h, N * sizeof(unsigned long long int), hipMemcpyHostToDevice);
	hipMemcpy(index_d, index_h, N * sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(x_d, x_h, N * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(y_d, y_h, N * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(z_d, z_h, N * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(vx_d, vx_h, N * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(vy_d, vy_h, N * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(vz_d, vz_h, N * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(A1_d, A1_h, N * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(A2_d, A2_h, N * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(A3_d, A3_h, N * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(jd_init_d, jd_init_h, N * sizeof(double), hipMemcpyHostToDevice);

}

__host__ void Host::Alloc2(){
	xt_h = (double*)malloc(N * sizeof(double));
	yt_h = (double*)malloc(N * sizeof(double));
	zt_h = (double*)malloc(N * sizeof(double));
	vxt_h = (double*)malloc(N * sizeof(double));
	vyt_h = (double*)malloc(N * sizeof(double));
	vzt_h = (double*)malloc(N * sizeof(double));

	kx_h = (double*)malloc(N * RKFn * sizeof(double));
	ky_h = (double*)malloc(N * RKFn * sizeof(double));
	kz_h = (double*)malloc(N * RKFn * sizeof(double));
	kvx_h = (double*)malloc(N * RKFn * sizeof(double));
	kvy_h = (double*)malloc(N * RKFn * sizeof(double));
	kvz_h = (double*)malloc(N * RKFn * sizeof(double));

	snew_h = (double2*)malloc(N * sizeof(double2));
	dtmin_h = (double*)malloc(N * sizeof(double));

	xTable_h = (double*)malloc(Nperturbers * RKFn * sizeof(double));
	yTable_h = (double*)malloc(Nperturbers * RKFn * sizeof(double));
	zTable_h = (double*)malloc(Nperturbers * RKFn * sizeof(double));

	if(useGPU > 0){
		hipMalloc((void **) &kx_d, N * RKFn * sizeof(double));
		hipMalloc((void **) &ky_d, N * RKFn * sizeof(double));
		hipMalloc((void **) &kz_d, N * RKFn * sizeof(double));
		hipMalloc((void **) &kvx_d, N * RKFn * sizeof(double));
		hipMalloc((void **) &kvy_d, N * RKFn * sizeof(double));
		hipMalloc((void **) &kvz_d, N * RKFn * sizeof(double));

		hipMalloc((void **) &snew_d, N * sizeof(double2));
		hipMalloc((void **) &scan_d, N * sizeof(int2));
		hipMalloc((void **) &N_d, sizeof(int));

		hipMalloc((void **) &xTable_d, Nperturbers * RKFn * sizeof(double));
		hipMalloc((void **) &yTable_d, Nperturbers * RKFn * sizeof(double));
		hipMalloc((void **) &zTable_d, Nperturbers * RKFn * sizeof(double));
	}

	a_h = (double*)malloc(RKFn * RKFn * sizeof(double));
	b_h = (double*)malloc(RKFn * sizeof(double));
	bb_h = (double*)malloc(RKFn * sizeof(double));
	c_h = (double*)malloc(RKFn * sizeof(double));

}
__host__ void Host::initialize2(){
	for(int i = 0; i < N; ++i){
		dtmin_h[i] = 1.0e6;
	}

	for(int i = 0; i < RKFn; ++i){
		for(int j = 0; j < RKFn; ++j){
			a_h[i * RKFn + j] = 0.0;
		}
		b_h[i] = 0.0;
		bb_h[i] = 0.0;
		c_h[i] = 0.0;
	}
}


__host__ void Host::initialize3(){
	for(int i = 0; i < N; ++i){
		dtmin_h[i] = 1.0e6;
	}

	//save coordinates for backward integrations
	for(int i = 0; i < N; ++i){
		x0_h[i] = x_h[i];
		y0_h[i] = y_h[i];
		z0_h[i] = z_h[i];
		vx0_h[i] = vx_h[i];
		vy0_h[i] = vy_h[i];
		vz0_h[i] = vz_h[i];
		A10_h[i] = A1_h[i];
		A20_h[i] = A2_h[i];
		A30_h[i] = A3_h[i];
		m0_h[i] = m_h[i];
		id0_h[i] = id_h[i];

		xb_h[i] = x_h[i];
		yb_h[i] = y_h[i];
		zb_h[i] = z_h[i];
		vxb_h[i] = vx_h[i];
		vyb_h[i] = vy_h[i];
		vzb_h[i] = vz_h[i];
		A1b_h[i] = A1_h[i];
		A2b_h[i] = A2_h[i];
		A3b_h[i] = A3_h[i];
		mb_h[i] = m_h[i];
		idb_h[i] = id_h[i];
//if(id_h[i] == 72057594038045489) printf("S %d %llu %.20g %.20g %.20g %.20g\n", i, id_h[i], m_h[i], x0_h[i], A1_h[i], snew_h[i].y);
	}
	if(useGPU > 0){
		hipMemcpy(m0_d, m0_h, N * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(id0_d, id0_h, N * sizeof(unsigned long long int), hipMemcpyHostToDevice);
		hipMemcpy(x0_d, x0_h, N * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(y0_d, y0_h, N * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(z0_d, z0_h, N * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(vx0_d, vx0_h, N * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(vy0_d, vy0_h, N * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(vz0_d, vz0_h, N * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(A10_d, A10_h, N * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(A20_d, A20_h, N * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(A30_d, A30_h, N * sizeof(double), hipMemcpyHostToDevice);
	}
}
__host__ void Host::restore3(){

	//restore coordinates for backward integrations
	for(int i = 0; i < N; ++i){
		dtmin_h[i] = 1.0e6;

		x_h[i] = xb_h[i];
		y_h[i] = yb_h[i];
		z_h[i] = zb_h[i];
		vx_h[i] = vxb_h[i];
		vy_h[i] = vyb_h[i];
		vz_h[i] = vzb_h[i];
		A1_h[i] = A1b_h[i];
		A2_h[i] = A2b_h[i];
		A3_h[i] = A3b_h[i];
		m_h[i] = mb_h[i];
		id_h[i] = idb_h[i];
		index_h[i] = i;

		x0_h[i] = x_h[i];
		y0_h[i] = y_h[i];
		z0_h[i] = z_h[i];
		vx0_h[i] = vx_h[i];
		vy0_h[i] = vy_h[i];
		vz0_h[i] = vz_h[i];
		A10_h[i] = A1_h[i];
		A20_h[i] = A2_h[i];
		A30_h[i] = A3_h[i];
		m0_h[i] = m_h[i];
		id0_h[i] = id_h[i];
		index0_h[i] = i;

//if(id_h[i] == 72057594038045489) printf("R %d %llu %.20g %.20g %.20g %.20g\n", i, id_h[i], m_h[i], x0_h[i], A1_h[i], snew_h[i].y);
	}
	if(useGPU > 0){
		hipMemcpy(m_d, m_h, N * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(id_d, id_h, N * sizeof(unsigned long long int), hipMemcpyHostToDevice);
		hipMemcpy(index_d, index_h, N * sizeof(unsigned int), hipMemcpyHostToDevice);
		hipMemcpy(x_d, x_h, N * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(y_d, y_h, N * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(z_d, z_h, N * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(vx_d, vx_h, N * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(vy_d, vy_h, N * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(vz_d, vz_h, N * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(A1_d, A1_h, N * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(A2_d, A2_h, N * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(A3_d, A3_h, N * sizeof(double), hipMemcpyHostToDevice);

		hipMemcpy(m0_d, m0_h, N * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(id0_d, id0_h, N * sizeof(unsigned long long int), hipMemcpyHostToDevice);
		hipMemcpy(index0_d, index0_h, N * sizeof(unsigned int), hipMemcpyHostToDevice);
		hipMemcpy(x0_d, x0_h, N * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(y0_d, y0_h, N * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(z0_d, z0_h, N * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(vx0_d, vx0_h, N * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(vy0_d, vy0_h, N * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(vz0_d, vz0_h, N * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(A10_d, A10_h, N * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(A20_d, A20_h, N * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(A30_d, A30_h, N * sizeof(double), hipMemcpyHostToDevice);
	}
}

__global__ void setSnew_kernel(double2 *snew_d, int N){

	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if(id < N){
		snew_d[id].x = 1.5;
		snew_d[id].y = 1.5;
	}
}

__host__ void Host::setSnew(){

	if(useGPU == 0){
		for(int i = 0; i < N; ++i){
			snew_h[i].x = 1.5;
			snew_h[i].y = 1.5;
		}
	}
	else{
		setSnew_kernel <<< (N + 255) / 256, 256 >>> (snew_d, N);
	}
}

__host__ void Host::reduce(int S){
	//reduce arrays for repeated integration with a smaller time step
	int k = Nperturbers;

	for(int i = Nperturbers; i < N; ++i){
		index0_h[i] = index_h[i]; 
	}
	
	for(int i = Nperturbers; i < N; ++i){
		if(snew_h[i].y < 1.0){
			int ii = index0_h[i];

			x_h[k] = x0_h[ii];
			y_h[k] = y0_h[ii];
			z_h[k] = z0_h[ii];
			vx_h[k] = vx0_h[ii];
			vy_h[k] = vy0_h[ii];
			vz_h[k] = vz0_h[ii];
			A1_h[k] = A10_h[ii];
			A2_h[k] = A20_h[ii];
			A3_h[k] = A30_h[ii];
			m_h[k] = m0_h[ii];
			id_h[k] = id0_h[ii];
			index_h[k] = index0_h[ii];

//printf("%d %d %u %llu\n", i, k, ii, id_h[k]);
printf("%d %d %u %llu %.20g %.20g\n", i, k, ii, id_h[k], x_h[k], A1_h[k]);
			++k; 
		}
	}
	N = k;

	runsN[S + 1] = N;
}
