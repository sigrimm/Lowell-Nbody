#include "hip/hip_runtime.h"
//for FIFO
#include <fcntl.h>
#include <sys/stat.h>
#include <sys/types.h>
#include <unistd.h>

//#include "define.h"
#include "Host.h"

#include "force.h"
#include "integrator.h"


int main(int argc, char*argv[]){


	Host H;

	int er = H.readparam(argc, argv);
	if(er <= 0){
		printf("Error in reading param.dat file\n");
		return 0;
	}
	printf("Reading param.dat file OK\n");

	//Erase Outbinary file
	if(H.outBinary == 1){
		if(H.outHelio == 1){
			sprintf(H.outfilename, "Outhelio.bin");
		}
		else{
			sprintf(H.outfilename, "Outbary.bin");
		}
		H.outfile = fopen(H.outfilename, "wb");
		fclose(H.outfile);
	}


	if(H.useFIFO == 2){	
		//sprintf(H.infilename, "210921_2148_genga_in_yarkovsky_elements.bin");
		//sprintf(H.infilename, "211208_1916_genga_in_2021-12-08_specific_desig.bin");
		//sprintf(H.infilename, "210801_2104_genga_in_GA.bin");
		//sprintf(H.infilename, "220301_2048_genga_in_new_last_14_days.bin");
		//sprintf(H.infilename, "220524_2256_genga_in_query_genga_input_40k.bin");
		//sprintf(H.infilename, "220524_2258_genga_in_query_genga_input_10k.bin");
	}
	
	//read console arguments for the binary file name
	//other arguments are checked later to overwright the head data	
	for(int i = 1; i < argc; i += 2){
		if(strcmp(argv[i], "-in") == 0){
			sprintf(H.infilename, "%s", argv[i + 1]);
		}
	}

	double timing[6];
	for(int i = 0; i < 6; ++i){
		timing[i] = 0.0;
	}
	float milliseconds = 0.0f;
	hipError_t error;

	hipEvent_t tt1;		//start time for timing
	hipEvent_t tt2;		//end time for timing
	hipEventCreate(&tt1);
	hipEventCreate(&tt2);

	hipEventRecord(tt1);

	// **************************************************
	// Read header or size of the initial conditions file
	// **************************************************

	if(H.useFIFO == 1){
		//This is only for testing
		printf("Error, useFIFO = 1 not supported.\n");
		return 0;

		const char *myfifo = "myfifo";
		const char *fifoCheck = "fifoCheck";
		// ###############################
		//create FIFO
		// ###############################
		int nn = 0;
		int fd;
		mkfifo(myfifo, 0666); //path, permission mode
		mkfifo(fifoCheck, 0666); //path, permission mode

		// ###############################
		// read N
		// ###############################
		fd = open(myfifo,O_RDONLY);
		read(fd, &nn, sizeof(int));
		close(fd);
		printf("fifo n: %d\n", nn);
		// ###############################
		// send back N to check
		// ###############################
		int fd1;
		fd1 = open(fifoCheck, O_WRONLY);	
		write(fd1, &nn, sizeof(int));
		close(fd1);
		printf("sent back\n");
	}
	

	if(H.useFIFO == 2){
		H.infile = fopen(H.infilename, "rb");

		if(H.infile == NULL){
			printf("Error, input file not found %s\n", H.infilename);
			return 0;
		}

		int NTP = 0;
		printf("read file\n");

		er = H.readHeader(H.infile, NTP);
		if(er == 0){
			return 0;
		}

		//H.NTP = 1;

		H.N += NTP;
	}

	if(H.useFIFO == 0){
		int n = H.readICSize();
		if(n == 0){
			printf("Error, reading initial conditions file failed\n");
			return 0;
		}
		H.N += n;
	}

	if(H.N >= 1024 * 1024){
		printf("Error, N is too large for scan kernels\n");
		return 0;
	}


	H.time = H.time0;
	// **************************************************
//H.time1 = 2451000.5;
//H.outInterval = 10.0;
//H.outStart = 2450800.5;

	//move this to parameter file
	H.dti = 2.0;



	//read console arguments
	for(int i = 1; i < argc; i += 2){
		if(strcmp(argv[i], "-Nsteps") == 0){
			H.Nsteps = atoll(argv[i + 1]);
		}
		else if(strcmp(argv[i], "-outInterval") == 0){
			H.outInterval = atoll(argv[i + 1]);
		}
		else if(strcmp(argv[i], "-outStart") == 0){
			H.outStart = atof(argv[i + 1]);
		}
		else if(strcmp(argv[i], "-dt") == 0){
			H.dti = atof(argv[i + 1]);
		}
		else if(strcmp(argv[i], "-dtMin") == 0){
			H.dtiMin[0] = atof(argv[i + 1]);
		}
		else if(strcmp(argv[i], "-endTime") == 0){
			H.time1 = atof(argv[i + 1]);
		}
		else if(strcmp(argv[i], "-useGPU") == 0){
			H.useGPU = atoi(argv[i + 1]);
		}
		else if(strcmp(argv[i], "-useFIFO") == 0){
			H.useFIFO = atoi(argv[i + 1]);
		}
		else if(strcmp(argv[i], "-outBinary") == 0){
			H.outBinary = atoi(argv[i + 1]);
		}
		else if(strcmp(argv[i], "-useAdaptive") == 0){
			H.useAdaptiveTimeSteps = atoi(argv[i + 1]);
		}
		else if(strcmp(argv[i], "-useIndividual") == 0){
			H.useIndividualTimeSteps = atoi(argv[i + 1]);
		}
		else if(strcmp(argv[i], "-NMax") == 0){
			H.NMax = atoi(argv[i + 1]);
		}
		else if(strcmp(argv[i], "-in") == 0){
			//this is already done
		}
		else{
			printf("Error, console argument not valid.\n");
			return 0;
		}
	}
	H.dt = H.dti * dayUnit;


	if(H.N - Nperturbers > H.NMax){
		printf("Number of bodies larger than Nmax\n");
		H.N = Nperturbers + H.NMax;
	}

	printf("infile name %s\n", H.infilename);

	printf("outStart: %.20g, time0: %.20g, time1: %.20g, outInterval: %lld\n", H.outStart, H.time0, H.time1, H.outInterval);

	printf("Nperturbers: %d N: %d\n", Nperturbers, H.N);


	// **************************************************
	// Allocate CPU and GPU memory
	// **************************************************
	H.Alloc1();

	hipDeviceSynchronize();
	error = hipGetLastError();
	printf("Alloc error = %d = %s\n",error, hipGetErrorString(error));
	if(error != 0.0){
		return 0;
	}

	// **************************************************

	H.initialize1();


	// **************************************************
	// Read initial conditions
	// **************************************************

	printf("Read initial conditions\n");
	if(H.useFIFO == 2){	
		//read particles
		er = H.readFile(H.infile);
		printf("read file OK\n");
		fclose(H.infile);

		/*						
		// -----------------------------------
		// Use this to extract a single object
		int ii = 29;//166;//29; //84;
		//int ii = 83;//166;//29; //84;
		H.N = Nperturbers + 1;
		int n = Nperturbers;

		H.id_h[n] = H.id_h[ii];
		H.x_h[n] = H.x_h[ii];
		H.y_h[n] = H.y_h[ii];
		H.z_h[n] = H.z_h[ii];
		H.vx_h[n] = H.vx_h[ii];
		H.vy_h[n] = H.vy_h[ii];
		H.vz_h[n] = H.vz_h[ii];
		H.A1_h[n] = H.A1_h[ii];
		H.A2_h[n] = H.A2_h[ii];
		H.A3_h[n] = H.A3_h[ii];
		H.jd_init_h[n] = H.jd_init_h[ii];
		
printf("xyz %.40g %.40g %.40g %.40g %.40g %.40g %.40g %.40g %.40g %.20g %llu\n", H.x_h[n], H.y_h[n], H.z_h[n], H.vx_h[n], H.vy_h[n], H.vz_h[n], H.A1_h[n], H.A2_h[n], H.A3_h[n], H.jd_init_h[n], H.id_h[n]);
		
		// -----------------------------------
		*/
		if(er == 0){
			return 0;
		}
	}
	if(H.useFIFO == 0){

		int er = H.Host::readIC();
		if(er == 0){
			return 0;
		}


	}

	//convert velocities and nonGrav terms
	H.convertV();
	// **************************************************

	// **************************************************
	// Read perturbers masses from perturbers.h file
	// **************************************************

	H.perturbersMass();
	H.perturbersIDs();
//m[Nperturbers] = 1.e-11; //ca mass of Flora

	// **************************************************
	//copy the data to the device
	if(H.useGPU > 0){
		H.copy1();
	}	
	// **************************************************

	// *******************************************************************
	// Allocate and set parameters for the Runge-Kutta-Fehlberg integrator
	// *******************************************************************
	H.Alloc2();
	H.initialize2();


	if(RKFn == 6){
		H.setRKF45();
	}
	else if(RKFn == 7){
		H.setDP54();
	}
	else if(RKFn == 13){
		H.setRKF78();
	}
	else{
		printf("RKFn values not valid %d\n", RKFn);
		return 0;
	}

	if(H.useGPU > 0){
		H.copyConst();
	}
	// *******************************************************************

	hipDeviceSynchronize();
	error = hipGetLastError();
	printf("copy error = %d = %s\n",error, hipGetErrorString(error));
	if(error != 0.0){
		return 0;
	}

	hipEventRecord(tt2);
	hipEventSynchronize(tt2);
	hipEventElapsedTime(&milliseconds, tt1, tt2);
	printf("Time for ic and allocation, %g seconds\n", milliseconds * 0.001);
	timing[0] += milliseconds * 0.001;


	hipEventRecord(tt1);


	// **************************************************


	// **************************************************
	//perturbers table
	// **************************************************
	
	er = H.readTable();
	if(er == 0){
		return 0;
	}
	// **************************************************

	error = hipGetLastError();
	printf("Perturbers error = %d = %s\n",error, hipGetErrorString(error));
	if(error != 0.0){
		return 0;
	}

	hipEventRecord(tt2);
	hipEventSynchronize(tt2);
	
	hipEventElapsedTime(&milliseconds, tt1, tt2);
	printf("Time for perturbers table, %g seconds\n", milliseconds * 0.001);
	timing[1] += milliseconds * 0.001;

	hipEventRecord(tt1);



	H.dtfile = fopen(H.dtfilename, "w");

	//###########################################
	// Start pre-integration
	//###########################################

	if(H.DoPreIntegration == 1){
		double dtiOld = H.dti;
		printf("dtiOld %g\n", dtiOld);

		er = H.preIntegration();
		if(er == 0){
			return 0;
		}
		H.dti = dtiOld;
		H.dt = H.dti * dayUnit;
		H.time = H.outStart;
	}
	else{
		H.time = H.time0;
	}

	//###########################################
	// End pre-integration
	//###########################################


	hipEventRecord(tt2);
	hipEventSynchronize(tt2);
	
	hipEventElapsedTime(&milliseconds, tt1, tt2);
	printf("Time for pre-integration, %g seconds\n", milliseconds * 0.001);
	timing[2] += milliseconds * 0.001;

	hipEventRecord(tt1);


	//save coordinates for repeated integrations
	H.initialize3();

	unsigned long long int outCount = 0;
	//###########################################
	// First output
	//###########################################

	H.output(outCount, H.time);
	++outCount;


	H.runsN[0] = H.N;

	double timeb = H.time;
	double time1b = H.time1;
	double outStartb = H.outStart;
	int N0 = H.N;


	//###########################################
	//loop for forward backward integration
	//###########################################
	//for(int b = 0; b < 1; ++b){	
	for(int b = 0; b < 2; ++b){	

		int DT0 = 2;
		int DT = DT0;
		unsigned long long int outI = H.outInterval;
		unsigned long long int cOut = 0llu;

		if(b == 0 && H.time1 <= H.time0){
			//skip forward integration
			continue;
		}


		if(b == 1){
			//backward integration
			H.restore3();
			H.time1 = outStartb;
			H.time = timeb;
			H.outStart = time1b;
			DT0 = -DT0;
			DT = -DT;
		}

		H.time0 = timeb;


		if(DT > 0 && H.outStart > H.time){
			outI = H.outStart - H.time;
		}
		if(DT < 0 && H.outStart < H.time){
			outI = H.time - H.outStart;
printf("outI %.20g %.20g\n", H.time, H.outStart);
		}


		//###########################################
		// Start time step loop
		//###########################################
		for(int tt = 0; tt < 100000; ++tt){


			if(cOut + fabs(DT) > outI && ((H.dt > 0 && H.time0 + H.dti >= H.outStart) || (H.dt < 0 && H.time + H.dti <= H.outStart))){
				DT = (outI - cOut);
				if(b == 1) DT = -DT;

printf("   correctT %.20g %.20g %d %llu %llu\n", H.time0, H.time0 + DT, DT, cOut, outI);
			}


printf("D %.20g %.20g %llu %llu %d\n", H.time0, H.time1, cOut, outI, DT);
			// loop for different time step ranges
			H.N = N0;
			H.save1();
			double time1 = H.time0 + DT;

			//###########################################
			// Loop over different time step levels
			//###########################################
			for(int S = 0; S < H.nRuns; ++S){
			//for(int S = 0; S < 1; ++S){

				H.dti = fabs(H.runsdt[S]);
				double dtmin = H.dti;

				if(fabs(H.dti) > fabs(DT)) H.dti = fabs(DT);

				//set dts, round dti down to power of 10
				{
					double l = log10(H.dti);
					double f = floor(l);
					double s = pow(10.0, f);
					H.dts = s * 0.1;
//printf("H.dti %g %g %g %g %g | %g %g\n", H.dti, l, f, s, H.dts, H.dti, H.dtiMin[S]);

					int dtt;
					if(H.dti >= 0.0){
					dtt = (fabs(H.dti) + (0.5 * H.dts)) / H.dts;
					}
					else{
					dtt = -(fabs(H.dti) - (0.5 * H.dts)) / H.dts;
					}
					H.dti = dtt * H.dts;
//printf("H.dtiB %.20g %d %g\n", H.dti, dtt, H.dts);


				}
				unsigned long long int Nci = (fabs(DT) + 0.5 * H.dts) / H.dts;

				if(b == 1){
					//backward integration
					H.dti = -H.dti;
				}


				H.time = H.time0;

				H.setSnew();


				//###########################################
				//Time step 
				//###########################################
				H.IntegrationLoop(S, Nci, time1, dtmin);

				hipDeviceSynchronize();

				hipEventRecord(tt2);
				hipEventSynchronize(tt2);
				
				hipEventElapsedTime(&milliseconds, tt1, tt2);
				timing[3 + S] += milliseconds * 0.001;

				printf("Time for integration %d, %g seconds\n", S + 1, timing[3 + S]);
				printf("With %d bodies\n", H.N);

				hipEventRecord(tt1);

				//###########################################
				// End time step
				//###########################################

				H.save(dtmin);

				//reduce arrays for repeated integration with a smaller time step
				if(H.useGPU > 0){
					H.reduceCall(S);
				}
				else{
					H.reduce(S);
				}

				if(H.N == Nperturbers){
					break;
				}

			} //end of S loop
			H.N = N0;
			H.time0 = H.time;
			cOut += fabs(DT);


			if(cOut >= outI && ((DT > 0 && H.time >= H.outStart) || (DT < 0 && H.time <= H.outStart))){
			//if(t % 10 == 0){

				H.output(outCount, H.time);
				++outCount;
				if(outCount >= 1e12){
					printf("Error, too many output files, %llu \n", outCount);
					return 0;
				}

				DT = DT0;
				cOut = 0llu;
				outI = H.outInterval; //needed only at the first time

			}


			if((DT > 0 && H.time >= H.time1) || (DT < 0 && H.time <= H.time1)){
				printf("Reached the end\n");
				break;
			}


		} // end of tt loop
		//###########################################
		// End time step loop
		//###########################################


	} // end of b loop

	fclose(H.dtfile);
	
	printf("Time for ic and allocation, %g seconds\n", timing[0]);
	printf("Time for perturbers table, %g seconds\n", timing[1]);
	printf("Time for pre-integration, %g seconds\n", timing[2]);
	printf("Time for integration 1, %g seconds\n", timing[3]);
	printf("Time for integration 2, %g seconds\n", timing[4]);
	printf("Time for integration 3, %g seconds\n", timing[5]);

	for(int i = 0; i < 4; ++i){
		printf("N %d %d %g\n", i, H.runsN[i] - Nperturbers, H.dtiMin[i]);
	}	

	FILE *timefile;
	timefile = fopen("Timing.dat", "a");
	//for(int i = 0; i < 4; ++i){
	for(int i = 0; i < 1; ++i){
		fprintf(timefile, "%d %d %g %g\n", i, H.runsN[i] - Nperturbers, H.dtiMin[i], timing[3 + i]);
	}
	fclose(timefile);
	
	return 0;

}
	
