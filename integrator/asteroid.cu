#include "asteroid.h"



int asteroid::allocateGPU(){

	hipMalloc((void **) &x_d, N * sizeof(double));
	hipMalloc((void **) &y_d, N * sizeof(double));
	hipMalloc((void **) &z_d, N * sizeof(double));

	hipMalloc((void **) &vx_d, N * sizeof(double));
	hipMalloc((void **) &vy_d, N * sizeof(double));
	hipMalloc((void **) &vz_d, N * sizeof(double));

	hipMalloc((void **) &xt_d, N * sizeof(double));
	hipMalloc((void **) &yt_d, N * sizeof(double));
	hipMalloc((void **) &zt_d, N * sizeof(double));

	hipMalloc((void **) &vxt_d, N * sizeof(double));
	hipMalloc((void **) &vyt_d, N * sizeof(double));
	hipMalloc((void **) &vzt_d, N * sizeof(double));

	hipMalloc((void **) &dx_d, N * sizeof(double));
	hipMalloc((void **) &dy_d, N * sizeof(double));
	hipMalloc((void **) &dz_d, N * sizeof(double));

	hipMalloc((void **) &dvx_d, N * sizeof(double));
	hipMalloc((void **) &dvy_d, N * sizeof(double));
	hipMalloc((void **) &dvz_d, N * sizeof(double));

	hipMalloc((void **) &kx_d, N * RKFn * sizeof(double));
	hipMalloc((void **) &ky_d, N * RKFn * sizeof(double));
	hipMalloc((void **) &kz_d, N * RKFn * sizeof(double));

	hipMalloc((void **) &kvx_d, N * RKFn * sizeof(double));
	hipMalloc((void **) &kvy_d, N * RKFn * sizeof(double));
	hipMalloc((void **) &kvz_d, N * RKFn * sizeof(double));

	hipMalloc((void **) &ax_d, N * sizeof(double));
	hipMalloc((void **) &ay_d, N * sizeof(double));
	hipMalloc((void **) &az_d, N * sizeof(double));

	hipMalloc((void **) &A1_d, N * sizeof(double));
	hipMalloc((void **) &A2_d, N * sizeof(double));
	hipMalloc((void **) &A3_d, N * sizeof(double));


	hipMalloc((void **) &a_d, RKFn * RKFn * sizeof(double));
	hipMalloc((void **) &b_d, RKFn * sizeof(double));
	hipMalloc((void **) &bb_d, RKFn * sizeof(double));
	hipMalloc((void **) &c_d, RKFn * sizeof(double));

	hipDeviceSynchronize();
	hipError_t error = hipGetLastError();
	printf("allocate error = %d = %s\n",error, hipGetErrorString(error));
	if(error != 0.0){
		return 0;
	}
	return 1;
}



int asteroid::copyIC(){

	hipMemcpy(x_d, x_h, N * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(y_d, y_h, N * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(z_d, z_h, N * sizeof(double), hipMemcpyHostToDevice);

	hipMemcpy(vx_d, vx_h, N * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(vy_d, vy_h, N * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(vz_d, vz_h, N * sizeof(double), hipMemcpyHostToDevice);

	hipMemcpy(A1_d, A1_h, N * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(A2_d, A2_h, N * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(A3_d, A3_h, N * sizeof(double), hipMemcpyHostToDevice);

	hipMemcpy(a_d, a_h, RKFn * RKFn * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(b_d, b_h, RKFn * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(bb_d, bb_h, RKFn * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(c_d, c_h, RKFn * sizeof(double), hipMemcpyHostToDevice);

	hipDeviceSynchronize();
	hipError_t error = hipGetLastError();
	printf("copy error = %d = %s\n",error, hipGetErrorString(error));
	if(error != 0.0){
		return 0;
	}

	return 1;
}

void asteroid::copyOutput(){

	hipMemcpy(x_h, x_d, N * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(y_h, y_d, N * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(z_h, z_d, N * sizeof(double), hipMemcpyDeviceToHost);

	hipMemcpy(vx_h, vx_d, N * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(vy_h, vy_d, N * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(vz_h, vz_d, N * sizeof(double), hipMemcpyDeviceToHost);

}

