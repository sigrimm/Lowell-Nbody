#include "asteroid.h"


int asteroid::allocateGPU(){

	//Check warp size
	hipDeviceProp_t devProp;
	int dev = 0;					//Set device number
	hipGetDeviceProperties(&devProp, dev);
	WarpSize = devProp.warpSize;


	hipMalloc((void **) &startTime_d, Nperturbers * RKFn * sizeof(double));
	hipMalloc((void **) &endTime_d, Nperturbers * RKFn * sizeof(double));
	hipMalloc((void **) &id_d, Nperturbers * sizeof(int));
	hipMalloc((void **) &nChebyshev_d, Nperturbers * sizeof(int));
	hipMalloc((void **) &offset0_d, Nperturbers * RKFn * sizeof(int));
	hipMalloc((void **) &offset1_d, Nperturbers * RKFn * sizeof(int));
	hipMalloc((void **) &GM_d, Nperturbers * sizeof(double));

	hipMalloc((void **) &cdata_d, Nperturbers * RKFn * nCm * 3 * sizeof(double));
	hipMalloc((void **) &data_d, datasize * sizeof(double));

	hipMalloc((void **) &xTable_d, Nperturbers * RKFn * sizeof(double));
	hipMalloc((void **) &yTable_d, Nperturbers * RKFn * sizeof(double));
	hipMalloc((void **) &zTable_d, Nperturbers * RKFn * sizeof(double));

	hipMalloc((void **) &vxTable_d, Nperturbers * RKFn * sizeof(double));
	hipMalloc((void **) &vyTable_d, Nperturbers * RKFn * sizeof(double));
	hipMalloc((void **) &vzTable_d, Nperturbers * RKFn * sizeof(double));

	hipMalloc((void **) &x_d, N * sizeof(double));
	hipMalloc((void **) &y_d, N * sizeof(double));
	hipMalloc((void **) &z_d, N * sizeof(double));

	hipMalloc((void **) &vx_d, N * sizeof(double));
	hipMalloc((void **) &vy_d, N * sizeof(double));
	hipMalloc((void **) &vz_d, N * sizeof(double));

	hipMalloc((void **) &dx_d, N * sizeof(double));
	hipMalloc((void **) &dy_d, N * sizeof(double));
	hipMalloc((void **) &dz_d, N * sizeof(double));

	hipMalloc((void **) &dvx_d, N * sizeof(double));
	hipMalloc((void **) &dvy_d, N * sizeof(double));
	hipMalloc((void **) &dvz_d, N * sizeof(double));

	hipMalloc((void **) &kx_d, N * RKFn * sizeof(double));
	hipMalloc((void **) &ky_d, N * RKFn * sizeof(double));
	hipMalloc((void **) &kz_d, N * RKFn * sizeof(double));

	hipMalloc((void **) &kvx_d, N * RKFn * sizeof(double));
	hipMalloc((void **) &kvy_d, N * RKFn * sizeof(double));
	hipMalloc((void **) &kvz_d, N * RKFn * sizeof(double));

	hipMalloc((void **) &ax_d, N * sizeof(double));
	hipMalloc((void **) &ay_d, N * sizeof(double));
	hipMalloc((void **) &az_d, N * sizeof(double));

	hipMalloc((void **) &A1_d, N * sizeof(double));
	hipMalloc((void **) &A2_d, N * sizeof(double));
	hipMalloc((void **) &A3_d, N * sizeof(double));


	hipMalloc((void **) &snew_d, N * sizeof(double));
	hipMalloc((void **) &ssum_d, N * sizeof(double));

	hipDeviceSynchronize();
	hipError_t error = hipGetLastError();
	printf("allocate error = %d = %s\n",error, hipGetErrorString(error));
	if(error != 0.0){
		return 0;
	}
	return 1;
}



int asteroid::readData(){

	int er;

	er = fread(data_h, sizeof(double), datasize, perturbersFile);

	if(er <= 0){
		return 0;
	}

	/*
	for(int i = 0; i < 20; ++i){
		printf("%.20g ", data_h[i]);
	}
	printf("\n");
	for(int i = datasize - 20; i < datasize; ++i){
		printf("%.20g ", data_h[i]);
	}
	printf("\n");
	*/
	
	return 1;
}


int asteroid::copyIC(){

	hipMemcpy(startTime_d, startTime_h, Nperturbers * RKFn * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(endTime_d, endTime_h, Nperturbers * RKFn * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(id_d, id_h, Nperturbers * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(nChebyshev_d, nChebyshev_h, Nperturbers * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(offset0_d, offset0_h, Nperturbers * RKFn * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(offset1_d, offset1_h, Nperturbers * RKFn * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(GM_d, GM_h, Nperturbers * sizeof(double), hipMemcpyHostToDevice);

	hipMemcpy(data_d, data_h, datasize * sizeof(double), hipMemcpyHostToDevice);

	hipMemcpy(x_d, x_h, N * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(y_d, y_h, N * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(z_d, z_h, N * sizeof(double), hipMemcpyHostToDevice);

	hipMemcpy(vx_d, vx_h, N * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(vy_d, vy_h, N * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(vz_d, vz_h, N * sizeof(double), hipMemcpyHostToDevice);

	hipMemcpy(A1_d, A1_h, N * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(A2_d, A2_h, N * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(A3_d, A3_h, N * sizeof(double), hipMemcpyHostToDevice);

	hipDeviceSynchronize();
	hipError_t error = hipGetLastError();
	printf("copy error = %d = %s\n",error, hipGetErrorString(error));
	if(error != 0.0){
		return 0;
	}

	return 1;
}

void asteroid::copyOutput(){

	hipMemcpy(x_h, x_d, N * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(y_h, y_d, N * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(z_h, z_d, N * sizeof(double), hipMemcpyDeviceToHost);

	hipMemcpy(vx_h, vx_d, N * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(vy_h, vy_d, N * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(vz_h, vz_d, N * sizeof(double), hipMemcpyDeviceToHost);

}

